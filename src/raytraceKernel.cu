#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>
#include "glm/glm.hpp"
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include <thrust/partition.h>
#include "glm/gtc/matrix_transform.hpp"
#include "glm/gtc/matrix_inverse.hpp"
#include "materials.h"


void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}


//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov,float focalDist,float aperture, int dof){
  
  ray r;
  float theta = fov.x*PI/180.0f;
  float phi = fov.y*PI/180.0f;

  glm::vec3 A = glm::cross(view,up);
  glm::vec3 B = glm::cross(A,view);
  glm::vec3 M = eye+view; 
  glm::vec3 H = glm::normalize(A)*glm::length(view)*tan(theta);
  glm::vec3 V = glm::normalize(B)*glm::length(view)*tan(phi);

  float sx= -1.0f; //(float)x/(resolution.x-1);
  float sy = -1.0f;//1.0 - (float)y/(resolution.y-1);
  
  thrust::default_random_engine rng(hash(43231*time));
  thrust::uniform_real_distribution<float> u01(-0.95,0.95);
  thrust::uniform_real_distribution<float> u02(-1.0,1.0);

  while( sx<=0.0f || sx>=1.0f || sx<=0.0f || sy>=1.0f)
	{
		float xrand = x + u01(rng);
		float yrand = y + u01(rng);
		sx = xrand/(resolution.x-1);
		sy = 1.0f - yrand/ (resolution.y-1);
	}

  glm::vec3 P = M + (2*sx-1)*H + (2*sy - 1)*V;
  
  r.origin = eye;
  r.direction = glm::normalize(P-r.origin);

  if(dof && focalDist!=0)
  {
	  float t = (focalDist-r.origin.z)/r.direction.z;
	  P = r.origin + t*r.direction;
	  glm::vec3 xaperture = aperture*H;
	  glm::vec3 yaperture = aperture*V;
	  r.origin= r.origin + u02(rng)*xaperture + u02(rng)*yaperture;
	  r.direction = glm::normalize(P-r.origin);
  }
  r.direction = glm::normalize(P-r.origin);
  return r;
}


//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec4* image){
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(index<= resolution.x*resolution.y){
      image[index] = glm::vec4(0,0,0,-1);
    }
}

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec4* image, ray* rays,int iterations){
  
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	int x = rays[index].pixelIndex.x;
	int y = rays[index].pixelIndex.y;
	int pixelIndex = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;    
      color.x = image[pixelIndex].x*255.0/iterations;
      color.y = image[pixelIndex].y*255.0/iterations;
      color.z = image[pixelIndex].z*255.0/iterations;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[pixelIndex].w = 0;
      PBOpos[pixelIndex].x = color.x;     
      PBOpos[pixelIndex].y = color.y;
      PBOpos[pixelIndex].z = color.z;

  }
}

__device__ bool isLight(int objId, int* lights, int numberOfLights)
{
	for (int i=0; i<numberOfLights; ++i)
		if (lights[i] == objId)
			return true;
	return false;
}


//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, float bounce, cameraData cam, int rayDepth, glm::vec4* colors, 
                            int* objidbuffer, staticGeom* geoms, int numberOfGeoms, material* materials, int numberOfMaterials, 
							map* maps, int numberOfMaps,int* lights, int numberOfLights,ray* rays,int dof, int* perlinData){

  //int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  //int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  //int index = x + (y * resolution.x);
	
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  int pixelIndex;
  int x=-1;
  int y=-1;
  ray r;

  if (bounce==1)
  {
	y = (int) (index/(int)resolution.x);
	x = (int) (index%(int)resolution.x);
	pixelIndex = index;
	r = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov,cam.focalDist,cam.aperture,dof);
	r.active = true;
	r.pixelIndex = glm::vec2(x,y);
	r.rayColor = glm::vec3(1,1,1);
	rays[index].rayColor = r.rayColor;
	rays[index].pixelIndex = r.pixelIndex;
  }
  else
  {
	  r = rays[index];
	  x = r.pixelIndex.x;
	  y = r.pixelIndex.y;
	  pixelIndex = x + (y*resolution.x);
  }

  if((x<=resolution.x && y<=resolution.y && r.active)){
	glm::vec3 intersectionPoint;
	glm::vec3 intersectionNormal;

	int objId = findNearestGeometricIntersection(r,intersectionPoint,intersectionNormal,geoms,numberOfGeoms);

	if (bounce==1 && time< 1.5f)
	{
		colors[pixelIndex].w = intersectionPoint.z;
		objidbuffer[pixelIndex] = objId;
	}
	if (objId == -1)
	{
		rays[index].active = false;
		rays[index].rayColor = glm::vec3(0,0,0);
		return;
	}
	material mtl = materials[geoms[objId].materialid];
	if (isLight(objId,lights,numberOfLights))
	{
		rays[index].active = false;
		rays[index].rayColor.x *= mtl.color.x*mtl.emittance;
		rays[index].rayColor.y *= mtl.color.y*mtl.emittance;
		rays[index].rayColor.z *= mtl.color.z*mtl.emittance;
		colors[pixelIndex].x += rays[index].rayColor.x;
		colors[pixelIndex].y += rays[index].rayColor.y;
		colors[pixelIndex].z += rays[index].rayColor.z;
		return;
	}

	glm::vec3 emittedColor;
	glm::vec3 unabsorbedColor;
	int bsdf = calculateBSDF(r,intersectionPoint,intersectionNormal,emittedColor,colors[index],unabsorbedColor,mtl,bounce*time*index);
	
	if (bsdf == 0)
	{
		glm::vec3 surfaceColor = getSurfaceColor(intersectionPoint,intersectionNormal,mtl,objId,geoms,maps,perlinData);
		r.rayColor.x *= surfaceColor.x;
		r.rayColor.y *= surfaceColor.y;
		r.rayColor.z *= surfaceColor.z;
	}
	else if(bsdf == 1)
	{
		r.rayColor.x *= mtl.specularColor.x;
		r.rayColor.y *= mtl.specularColor.y;
		r.rayColor.z *= mtl.specularColor.z;
	}
	else if (bsdf == 2)
	{
		glm::vec3 surfaceColor = getSurfaceColor(intersectionPoint,intersectionNormal,mtl,objId,geoms,maps,perlinData);
		r.rayColor.x *= surfaceColor.x;
		r.rayColor.y *= surfaceColor.y;
		r.rayColor.z *= surfaceColor.z;
	}

	rays[index].origin = r.origin + 0.001f*r.direction;
	rays[index].direction = r.direction;
	rays[index].active = r.active;
	rays[index].pixelIndex = r.pixelIndex;
	rays[index].rayColor = r.rayColor;
   }
	
}

__global__ void createBinaryActiveArray(glm::vec2 resolution,ray* rays,int* activeRaysArray, int* dNumActive)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < *dNumActive)
	{
		activeRaysArray[index] = rays[index].active?1:0;
	}
}

__global__ void parallelScanOnGPU(int* cudaActiveArray, int* gpuParallelScanTempArray, int* dNumActive,int d)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	int exponent = powf(2,d-1);
	if (index< (*dNumActive) )
	{
	if( index>= exponent)
		gpuParallelScanTempArray[index] = cudaActiveArray[index-exponent] + cudaActiveArray[index];
	else
		gpuParallelScanTempArray[index] = cudaActiveArray[index];
	}

}

__host__ void parallelScanActiveArray(int* cudaActiveArray,int* gpuParallelScanTempArray,int* hNumActive,int* dNumActive)
{
	int numberOfThreads = *hNumActive;
	float logN  = logf(numberOfThreads);
	int tileSize = 8;
	dim3 threadsPerBlock(tileSize*tileSize);
	dim3 fullBlocksPerGrid ( (int) ceil( (float)numberOfThreads/(tileSize*tileSize)));
	int* cudaActiveArrayCopy = NULL;
	hipMalloc((void**)&cudaActiveArrayCopy,numberOfThreads*sizeof(int));
	hipMemcpy(cudaActiveArrayCopy,cudaActiveArray,numberOfThreads*sizeof(int),hipMemcpyDeviceToDevice);
	int* activeArray = cudaActiveArrayCopy;
	int* tempArray = gpuParallelScanTempArray;
	int dmax = ceil(logN)+3; 

	for(int d=1; d<= dmax; ++d)
	{
	  parallelScanOnGPU<<<fullBlocksPerGrid, threadsPerBlock>>>(activeArray,tempArray,dNumActive,d);
	  int *swapTemp = activeArray;
	  activeArray = tempArray;
	  tempArray = swapTemp;
	}	
	
	if (tempArray != gpuParallelScanTempArray)
		hipMemcpy(activeArray,tempArray,numberOfThreads*sizeof(int),hipMemcpyDeviceToDevice);

	hipMemcpy(dNumActive,&gpuParallelScanTempArray[numberOfThreads-1],sizeof(int),hipMemcpyDeviceToDevice);

	hipFree(cudaActiveArrayCopy);
}

__global__ void streamCompact(ray* rays, ray* tempRays, int* rayActiveArray, int* scannedArray,int* dNumActive)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < *dNumActive && rayActiveArray[index])
	{
		rays[scannedArray[index]-1] = tempRays[index];
	}
}

__global__ void duplicateRaysArray(ray* dest, ray* src, int* dNumActive)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < *dNumActive)
	{
		dest[index] = src[index];
	}
}

struct is_not_active
{
	__device__ bool operator() (const ray r)
	{
		return !r.active;
	}
};

struct is_active
{
	__device__ bool operator() (const ray r)
	{
		return r.active;
	}
};


__global__ void moveWorld( staticGeom* geoms, staticGeom* prevGeoms, float t,int numberOfGeoms)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if(index < numberOfGeoms)
	{
		t = 1 - t*t;
		glm::vec3 newTranslation = t*geoms[index].translation + (1-t)*prevGeoms[index].translation;
		glm::mat4 translationMat = glm::translate(glm::mat4(), newTranslation);
		glm::mat4 rotationMat = glm::rotate(glm::mat4(), geoms[index].rotation.x, glm::vec3(1,0,0));
		rotationMat = rotationMat*glm::rotate(glm::mat4(), geoms[index].rotation.y, glm::vec3(0,1,0));
		rotationMat = rotationMat*glm::rotate(glm::mat4(), geoms[index].rotation.z, glm::vec3(0,0,1));
		glm::mat4 scaleMat = glm::scale(glm::mat4(), geoms[index].scale);
		glm::mat4 a =  translationMat*rotationMat*scaleMat;
		cudaMat4 m; 
		glm::mat4 b = glm::transpose(a);
		m.x = b[0];
		m.y = b[1];
		m.z = b[2];
		m.w = b[3];
		geoms[index].transform = m;

		a = glm::transpose(glm::inverse(a));
		m.x = a[0];
		m.y = a[1];
		m.z = a[2];
		m.w = a[3];
		geoms[index].inverseTransform = m;
	}
}

//TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials,map* maps,int numberOfMaps, geom* geoms, int numberOfGeoms,int mblur,int dof){
  
  int traceDepth = 1; //determines how many bounces the raytracer traces
  int perlinNumbers[512] = { 151,160,137,91,90,15,
  131,13,201,95,96,53,194,233,7,225,140,36,103,30,69,142,8,99,37,240,21,10,23,
  190,6,148,247,120,234,75,0,26,197,62,94,252,219,203,117,35,11,32,57,177,33,
  88,237,149,56,87,174,20,125,136,171,168,68,175,74,165,71,134,139,48,27,166,
  77,146,158,231,83,111,229,122,60,211,133,230,220,105,92,41,55,46,245,40,244,
  102,143,54, 65,25,63,161, 1,216,80,73,209,76,132,187,208,89,18,169,200,196,
  135,130,116,188,159,86,164,100,109,198,173,186,3,64,52,217,226,250,124,123,
  5,202,38,147,118,126,255,82,85,212,207,206,59,227,47,16,58,17,182,189,28,42,
  23,183,170,213,119,248,152, 2,44,154,163, 70,221,153,101,155,167,43,172,9,
  129,22,39,253,19,98,108,110,79,113,224,232,178,185, 112,104,218,246,97,228,
  251,34,242,193,238,210,144,12,191,179,162,241,81,51,145,235,249,14,239,107,
  49,192,214, 31,181,199,106,157,184, 84,204,176,115,121,50,45,127,4,150,254,
  138,236,205,93,222,114,67,29,24,72,243,141,128,195,78,66,215,61,156,180,
  151,160,137,91,90,15,
  131,13,201,95,96,53,194,233,7,225,140,36,103,30,69,142,8,99,37,240,21,10,23,
  190,6,148,247,120,234,75,0,26,197,62,94,252,219,203,117,35,11,32,57,177,33,
  88,237,149,56,87,174,20,125,136,171,168,68,175,74,165,71,134,139,48,27,166,
  77,146,158,231,83,111,229,122,60,211,133,230,220,105,92,41,55,46,245,40,244,
  102,143,54, 65,25,63,161, 1,216,80,73,209,76,132,187,208,89,18,169,200,196,
  135,130,116,188,159,86,164,100,109,198,173,186,3,64,52,217,226,250,124,123,
  5,202,38,147,118,126,255,82,85,212,207,206,59,227,47,16,58,17,182,189,28,42,
  23,183,170,213,119,248,152, 2,44,154,163, 70,221,153,101,155,167,43,172,9,
  129,22,39,253,19,98,108,110,79,113,224,232,178,185, 112,104,218,246,97,228,
  251,34,242,193,238,210,144,12,191,179,162,241,81,51,145,235,249,14,239,107,
  49,192,214, 31,181,199,106,157,184, 84,204,176,115,121,50,45,127,4,150,254,
  138,236,205,93,222,114,67,29,24,72,243,141,128,195,78,66,215,61,156,180
  };


  // set up crucial magic
  int numberOfThreads = (int)(renderCam->resolution.x)*(int)(renderCam->resolution.y);
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize*tileSize);
  dim3 fullBlocksPerGrid ( (int) ceil( (float)numberOfThreads/(tileSize*tileSize)));

  //send image to GPU
  glm::vec4* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec4));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec4), hipMemcpyHostToDevice);

  int* cudaobjidbuffer = NULL;
  hipMalloc((void**)&cudaobjidbuffer, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(int));
  
   //package lights
  std::vector<int> lightsVector;

  ray* cudarays = NULL;
  hipMalloc((void**)&cudarays, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(ray));

  int* cudaperlindata = NULL;
  hipMalloc((void**)&cudaperlindata, 512*sizeof(int));
  hipMemcpy( cudaperlindata, perlinNumbers, 512*sizeof(int), hipMemcpyHostToDevice);

  /*ALL STUFF FOR MANUAL STREAM COMPACTION
  ray* cudatemprays = NULL;
  hipMalloc((void**)&cudatemprays, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(ray));

  ////TEST SCAN
  //const int testNum = 2048;
  //int test[testNum];
  //for(int i=0; i<testNum;++i)
  //{
	 // test[i] = 1;
  //}

  int* cudaActiveArray = NULL;
  hipMalloc((void**)&cudaActiveArray, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(int));
  
  ////TEST SCAN
  //hipMalloc((void**)&cudaActiveArray, testNum*sizeof(int));
  //hipMemcpy( cudaActiveArray,test, testNum*sizeof(int), hipMemcpyHostToDevice);

  int* dParallelScanTempArray = NULL;
  hipMalloc((void**)&dParallelScanTempArray, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(int));

  ////TEST SCAN
  //hipMalloc((void**)&dParallelScanTempArray, testNum*sizeof(int));

  ////TEST SCAN
  //numberOfThreads = testNum;

  int* dNumActiveRays = NULL;
  hipMalloc((void**)&dNumActiveRays,sizeof(int));
  hipMemcpy( dNumActiveRays,&numberOfThreads, sizeof(int), hipMemcpyHostToDevice);
  */

  //package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  staticGeom* geomListPrevFrame = new staticGeom[numberOfGeoms];

  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
	staticGeom prevStaticGeom;
	int prevFrame = frame-1;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;

	if (frame==0)
	{
		prevFrame = frame;
	}
    prevStaticGeom.type = geoms[i].type;
    prevStaticGeom.materialid = geoms[i].materialid;
    prevStaticGeom.translation = geoms[i].translations[prevFrame];
    prevStaticGeom.rotation = geoms[i].rotations[prevFrame];
    prevStaticGeom.scale = geoms[i].scales[prevFrame];
    prevStaticGeom.transform = geoms[i].transforms[prevFrame];
    prevStaticGeom.inverseTransform = geoms[i].inverseTransforms[prevFrame];
    geomListPrevFrame[i] = prevStaticGeom;

	if (materials[geoms[i].materialid].emittance > 0.0f)
		lightsVector.push_back(i);
  }
  

  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);
 
  staticGeom* cudageomsPrevFrame = NULL;
  hipMalloc((void**)&cudageomsPrevFrame, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageomsPrevFrame, geomListPrevFrame, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);

  material* cudamaterials = NULL;
  hipMalloc((void**)&cudamaterials, numberOfMaterials*sizeof(material));
  hipMemcpy( cudamaterials, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

  map* cudamaps = NULL;
  hipMalloc((void**)&cudamaps, numberOfMaps*sizeof(map));
  hipMemcpy( cudamaps, maps, numberOfMaps*sizeof(map), hipMemcpyHostToDevice);


  int numberOfLights = lightsVector.size();
  int* cudalights = NULL;
  hipMalloc( (void**)&cudalights, numberOfLights*sizeof(int));
  hipMemcpy(cudalights,&lightsVector[0],numberOfLights*sizeof(int),hipMemcpyHostToDevice);

  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;
  cam.aperture = renderCam->aperture;
  cam.focalDist = renderCam->focalDist;

 /*TESTING STREAM COMPACTION
  int t = numberOfThreads;
  for(int k=1; k<=t; k++)
  {
  hipMemcpy( dNumActiveRays, &k, sizeof(int),hipMemcpyHostToDevice);
  parallelScanActiveArray(cudaActiveArray,dParallelScanTempArray,&k,dNumActiveRays);
  hipMemcpy(&numberOfThreads,dNumActiveRays,sizeof(int),hipMemcpyDeviceToHost);
  std::cout<<"NUM OF ACTIVE THREADS: "<<numberOfThreads<<std::endl;
  if (k == t)
  {
  int* resultArray = new int[k];
  hipMemcpy(resultArray,cudaActiveArray,k*sizeof(int),hipMemcpyDeviceToHost);
  for(int i=0; i<k;++i)
	  std::cout<<resultArray[i]<<" ";
  std::cout<<std::endl;
    hipMemcpy(resultArray,dParallelScanTempArray,k*sizeof(int),hipMemcpyDeviceToHost);
  for(int i=0; i<k;++i)
	  std::cout<<resultArray[i]<<" ";
  std::cout<<std::endl;
  
  delete[] resultArray;
  }
  }*/
   
  //Prepare scene for motion blur
 if(mblur)
 {
	 dim3 mblurThreadsPerBlock(numberOfGeoms);
	 dim3 mblurBlocksPerGrid( (int) ceil( (float)numberOfGeoms/(tileSize*tileSize)));
	 thrust::default_random_engine rng(hash(48589.0f*iterations));
     thrust::uniform_real_distribution<float> u01(0,1);
	 float interpolant = u01(rng);

	 moveWorld<<<mblurBlocksPerGrid,mblurThreadsPerBlock>>>(cudageoms, cudageomsPrevFrame, interpolant,numberOfGeoms);
 }

  thrust::device_ptr<ray> thrustRaysArray = thrust::device_pointer_cast(cudarays);
 
  //kernel launches
  for(int bounce = 1; bounce <= 8; ++bounce)
  {
	dim3 compactedBlocksPerGrid ( (int) ceil( (float)numberOfThreads/(tileSize*tileSize)));
	raytraceRay<<<compactedBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, (float)bounce, cam, traceDepth, cudaimage,cudaobjidbuffer, cudageoms, numberOfGeoms, cudamaterials, numberOfMaterials,cudamaps,numberOfMaps, cudalights,numberOfLights,cudarays,dof,cudaperlindata);
	numberOfThreads = thrust::partition(thrustRaysArray,thrustRaysArray+numberOfThreads,is_active()) - thrustRaysArray;
	//numberOfThreads = thrust::remove_if(thrustRaysArray,thrustRaysArray+numberOfThreads,is_not_active()) - thrustRaysArray;

	//MANUAL STREAM COMPACTION ATTEMPT
	//duplicateRaysArray<<<compactedBlocksPerGrid, threadsPerBlock>>>(cudatemprays,cudarays,dNumActiveRays);
	//createBinaryActiveArray<<<compactedBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution,cudarays,cudaActiveArray,dNumActiveRays);
	//thrust::inclusive_scan(thrustActiveArray,thrustActiveArray+numberOfThreads,thrustTempScanArray);
	//streamCompact<<<compactedBlocksPerGrid, threadsPerBlock>>>(cudarays,cudatemprays,cudaActiveArray,dParallelScanTempArray,dNumActiveRays);
	//hipMemcpy(dNumActiveRays,&dParallelScanTempArray[numberOfThreads-1],sizeof(int),hipMemcpyDeviceToDevice);
	//numberOfThreads = thrustTempScanArray[numberOfThreads-1];
	//parallelScanActiveArray(cudaActiveArray,dParallelScanTempArray,&numberOfThreads,dNumActiveRays);
	//hipMemcpy(&numberOfThreads,dNumActiveRays,sizeof(int),hipMemcpyDeviceToHost);
	//streamCompact<<<compactedBlocksPerGrid, threadsPerBlock>>>(cudarays,cudatemprays,cudaActiveArray,dParallelScanTempArray,dNumActiveRays);
  }

  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage,cudarays,iterations);
  


  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec4), hipMemcpyDeviceToHost);
  if(iterations == 1)
	hipMemcpy( renderCam->objIdBuffer, cudaobjidbuffer, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(int), hipMemcpyDeviceToHost);

  //free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  hipFree( cudamaterials );
  hipFree(cudamaps);
  hipFree(cudalights);
  hipFree(cudarays);
  /*CLEANUP FOR MANUAL STREAM COMPACTION DATA
  hipFree(cudatemprays);
  hipFree(cudaActiveArray);
  hipFree(dParallelScanTempArray);
  hipFree(dNumActiveRays);*/
  hipFree(cudaobjidbuffer);
  hipFree(cudaperlindata);
  delete [] geomList;
  delete [] geomListPrevFrame;

  // make certain the kernel has completed 
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}

/* THE CODE BELOW THIS IS FOR FUNCTIONS THAT RUN IN "TEXTURE" MODE. 
THIS CODE IS DUPLICATED TO AVOID UNNECESSARY BRANCHING IN THE GPU*/

//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRayT(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec4* colors, 
                            int* objidbuffer, staticGeom* geoms, int numberOfGeoms, material* materials, int numberOfMaterials, 
							map* maps, int numberOfMaps,int* lights, int numberOfLights,ray* rays,int dof, int* perlinData){

  //int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  //int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  //int index = x + (y * resolution.x);
	
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	int y = (int) (index/(int)resolution.x);
	int x = (int) (index%(int)resolution.x);
	int pixelIndex = index;
	ray r = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov,cam.focalDist,cam.aperture,dof);  
	rays[index].pixelIndex = glm::vec2(x,y);

  if(x<=resolution.x && y<=resolution.y){
	glm::vec3 intersectionPoint;
	glm::vec3 intersectionNormal;

	int objId = findNearestGeometricIntersection(r,intersectionPoint,intersectionNormal,geoms,numberOfGeoms);

	if (time< 1.5f)
	{
		colors[pixelIndex].w = intersectionPoint.z;
		objidbuffer[pixelIndex] = objId;
	}
	if (objId == -1)
	{
		return;
	}
	material mtl = materials[geoms[objId].materialid];
	if (isLight(objId,lights,numberOfLights))
	{
		colors[pixelIndex].x += mtl.color.x*mtl.emittance;
		colors[pixelIndex].y += mtl.color.y*mtl.emittance;
		colors[pixelIndex].z += mtl.color.z*mtl.emittance;
		return;
	}
	
	glm::vec3 surfaceColor = getSurfaceColor(intersectionPoint,intersectionNormal,mtl,objId,geoms,maps,perlinData);
	colors[pixelIndex].x += surfaceColor.x;
	colors[pixelIndex].y += surfaceColor.y;
	colors[pixelIndex].z += surfaceColor.z;
}
  }

//TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCoreT(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials,map* maps,int numberOfMaps, geom* geoms, int numberOfGeoms,int mblur,int dof){
  
  int traceDepth = 1;
  int perlinNumbers[512] = { 151,160,137,91,90,15,
  131,13,201,95,96,53,194,233,7,225,140,36,103,30,69,142,8,99,37,240,21,10,23,
  190,6,148,247,120,234,75,0,26,197,62,94,252,219,203,117,35,11,32,57,177,33,
  88,237,149,56,87,174,20,125,136,171,168,68,175,74,165,71,134,139,48,27,166,
  77,146,158,231,83,111,229,122,60,211,133,230,220,105,92,41,55,46,245,40,244,
  102,143,54, 65,25,63,161, 1,216,80,73,209,76,132,187,208,89,18,169,200,196,
  135,130,116,188,159,86,164,100,109,198,173,186,3,64,52,217,226,250,124,123,
  5,202,38,147,118,126,255,82,85,212,207,206,59,227,47,16,58,17,182,189,28,42,
  23,183,170,213,119,248,152, 2,44,154,163, 70,221,153,101,155,167,43,172,9,
  129,22,39,253,19,98,108,110,79,113,224,232,178,185, 112,104,218,246,97,228,
  251,34,242,193,238,210,144,12,191,179,162,241,81,51,145,235,249,14,239,107,
  49,192,214, 31,181,199,106,157,184, 84,204,176,115,121,50,45,127,4,150,254,
  138,236,205,93,222,114,67,29,24,72,243,141,128,195,78,66,215,61,156,180,
  151,160,137,91,90,15,
  131,13,201,95,96,53,194,233,7,225,140,36,103,30,69,142,8,99,37,240,21,10,23,
  190,6,148,247,120,234,75,0,26,197,62,94,252,219,203,117,35,11,32,57,177,33,
  88,237,149,56,87,174,20,125,136,171,168,68,175,74,165,71,134,139,48,27,166,
  77,146,158,231,83,111,229,122,60,211,133,230,220,105,92,41,55,46,245,40,244,
  102,143,54, 65,25,63,161, 1,216,80,73,209,76,132,187,208,89,18,169,200,196,
  135,130,116,188,159,86,164,100,109,198,173,186,3,64,52,217,226,250,124,123,
  5,202,38,147,118,126,255,82,85,212,207,206,59,227,47,16,58,17,182,189,28,42,
  23,183,170,213,119,248,152, 2,44,154,163, 70,221,153,101,155,167,43,172,9,
  129,22,39,253,19,98,108,110,79,113,224,232,178,185, 112,104,218,246,97,228,
  251,34,242,193,238,210,144,12,191,179,162,241,81,51,145,235,249,14,239,107,
  49,192,214, 31,181,199,106,157,184, 84,204,176,115,121,50,45,127,4,150,254,
  138,236,205,93,222,114,67,29,24,72,243,141,128,195,78,66,215,61,156,180
  };


  // set up crucial magic
  int numberOfThreads = (int)(renderCam->resolution.x)*(int)(renderCam->resolution.y);
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize*tileSize);
  dim3 fullBlocksPerGrid ( (int) ceil( (float)numberOfThreads/(tileSize*tileSize)));

  //send image to GPU
  glm::vec4* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec4));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec4), hipMemcpyHostToDevice);

  int* cudaobjidbuffer = NULL;
  hipMalloc((void**)&cudaobjidbuffer, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(int));
  
   //package lights
  std::vector<int> lightsVector;

  ray* cudarays = NULL;
  hipMalloc((void**)&cudarays, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(ray));

  int* cudaperlindata = NULL;
  hipMalloc((void**)&cudaperlindata, 512*sizeof(int));
  hipMemcpy( cudaperlindata, perlinNumbers, 512*sizeof(int), hipMemcpyHostToDevice);


  //package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];

  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;


	if (materials[geoms[i].materialid].emittance > 0.0f)
		lightsVector.push_back(i);
  }
  

  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);
 

  material* cudamaterials = NULL;
  hipMalloc((void**)&cudamaterials, numberOfMaterials*sizeof(material));
  hipMemcpy( cudamaterials, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

  map* cudamaps = NULL;
  hipMalloc((void**)&cudamaps, numberOfMaps*sizeof(map));
  hipMemcpy( cudamaps, maps, numberOfMaps*sizeof(map), hipMemcpyHostToDevice);


  int numberOfLights = lightsVector.size();
  int* cudalights = NULL;
  hipMalloc( (void**)&cudalights, numberOfLights*sizeof(int));
  hipMemcpy(cudalights,&lightsVector[0],numberOfLights*sizeof(int),hipMemcpyHostToDevice);

  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;
  cam.aperture = renderCam->aperture;
  cam.focalDist = renderCam->focalDist;

 
  //kernel launches

  dim3 compactedBlocksPerGrid ( (int) ceil( (float)numberOfThreads/(tileSize*tileSize)));
  raytraceRayT<<<compactedBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage,cudaobjidbuffer, cudageoms, numberOfGeoms, cudamaterials, numberOfMaterials,cudamaps,numberOfMaps, cudalights,numberOfLights,cudarays,dof,cudaperlindata);

  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage,cudarays,iterations);
  
  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec4), hipMemcpyDeviceToHost);
  if(iterations == 1)
	hipMemcpy( renderCam->objIdBuffer, cudaobjidbuffer, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(int), hipMemcpyDeviceToHost);

  //free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  hipFree( cudamaterials );
  hipFree(cudamaps);
  hipFree(cudalights);
  hipFree(cudarays);
  hipFree(cudaobjidbuffer);
  hipFree(cudaperlindata);
  delete [] geomList;

  // make certain the kernel has completed 
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}
