#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif


//global variables
rayBounce* cudaFirstPass;
rayBounce* cudaRayPool;		//for stream compaction, pool of rays that are still alive
rayBounce* cudaTempRayPool;	//for switching and replacing rays in stream compaction
int* cudaCompactA;
int* cudaCompactB;

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){
	
	ray r;

	//ray creation from camear stuff to be used in raycastFromCameraKernel
	glm::vec3 M = eye + view;	//center of screen

	//project screen to world space
	glm::vec3 A = glm::cross(view, up);
	glm::vec3 B = glm::cross(A, view);

	float C = glm::length(view);

	float phi = fov.y/180.0f * PI;		//convert to radians
	B = glm::normalize(B);
	glm::vec3 V = C * tan(phi) * B;

	float theta = fov.x/180.0f * PI;
	A = glm::normalize(A);
	glm::vec3 H = C * tan(theta) * A;

	//find the world space coord of the pixel
	float sx = (float)x / (resolution.x-1.0f);
	float sy = (float)y / (resolution.y-1.0f);

	glm::vec3 P = M + H * (2.0f * sx - 1.0f) + V * (1.0f - 2.0f * sy);

	r.origin = eye;
	r.direction = glm::normalize(P - eye);

	return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image, float iterations){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  //iterations = 1;

  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;
      color.x = image[index].x*255.0 / iterations;
      color.y = image[index].y*255.0 / iterations;
      color.z = image[index].z*255.0 / iterations;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}


//Does intersection on all of the objects and returns length of closest intersection
__host__ __device__ float testGeomIntersection(staticGeom* geoms, int numberOfGeoms, ray& r, glm::vec3& intersectionPoint, glm::vec3& normal, int& objID){

	float len = FLT_MAX;	
	float tempLen = -1;
	glm::vec3 tempIntersection;
	glm::vec3 tempNormal;
	
	//check for interesction
	for(int geomInd = 0; geomInd<numberOfGeoms; ++geomInd){
			
		if(geoms[geomInd].type == CUBE){
			tempLen = boxIntersectionTest(geoms[geomInd], r, tempIntersection, tempNormal);
		}

		else if (geoms[geomInd].type == SPHERE){
			tempLen = sphereIntersectionTest(geoms[geomInd], r, tempIntersection, tempNormal);
		}
			
		else if(geoms[geomInd].type == MESH){
				
		}
							
		//if intersection occurs and object is in front of previously intersected object
		if(tempLen != -1 && tempLen < len){
			len =tempLen;
			intersectionPoint = tempIntersection;
			normal = tempNormal;
			objID = geomInd;
		}
	}

	return len;

}

__global__ void streamCompact(int numRays, int* compactIn, int* compactOut, int maxDepth, int d){

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < numRays){
		
		if( index >= d){
			compactOut[index] = compactIn[index - d] + compactIn[index];
		}
		else{
			compactOut[index] = compactIn[index];
		}
	}

	__syncthreads(); 
}


__global__ void shiftRight(int* compactIn, int* compactOut, int numRays){

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if(index == 0)
		compactOut[0] = 0;
	else if (index < numRays)
		compactOut[index] = compactIn[index - 1];

}


__global__ void buildRayPool(int* compactIn, rayBounce* rayTempPass, rayBounce* rayPass, int numRays){

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < numRays){
		if(!rayTempPass[index].dead)
			rayPass[index] = rayTempPass[compactIn[index]];
	}

}

//creates and stores first bounce rays, always at depth 1
__global__ void createRay(glm::vec2 resolution, cameraData cam, staticGeom* geoms, int numberOfGeoms, material* materials, 
						 int numLights, int* lightID, rayBounce* firstPass, int maxDepth, int* compactIn, int numRays, glm::vec3* colors){

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * cam.resolution.x);
	
	glm::vec3 intersection;
	glm::vec3 normal;

	if(index < numRays){

		ray firstRay = raycastFromCameraKernel(resolution, x, y, cam.position, cam.view, cam.up, cam.fov); 
		
		//DOF and antialiasing setup
		//float focalLength = cam.focalLength;
		//float aperture = cam.aperture;
		//
		//glm::vec3 focalPoint = firstRay.origin + focalLength * firstRay.direction;

		////jitter camera
		//glm::vec3 jitterVal = 2.0f * aperture * generateRandomNumberFromThread(resolution, time, x, y);
		//jitterVal -= glm::vec3(aperture);
		//firstRay.origin += jitterVal;

		////find new direction
		//firstRay.direction = glm::normalize(focalPoint - firstRay.origin);

		////antialias sample per pixel
		//jitterVal = generateRandomNumberFromThread(resolution, time, x, y);
		//jitterVal -= glm::vec3(0.5f, 0.5f, 0.5f);
		//firstRay.direction += 0.0015f* jitterVal; 

		//do intersection test
		int objID = -1;
		float len = testGeomIntersection(geoms, numberOfGeoms, firstRay, intersection, normal, objID);
			
		//if no intersection, return
		if(objID == -1){
			firstPass[index] = rayBounce();
			firstPass[index].dead = true;
			firstPass[index].pixID = index;
			compactIn[index] = 0;
			return;	
		}
		
		int matID = geoms[objID].materialid;
		
		//save the first bounce information
		if(materials[matID].hasReflective == 1){
			firstPass[index] = rayBounce();
			firstPass[index].intersectPt = intersection;
			firstPass[index].normal = normal;
			firstPass[index].matID = matID;
			firstPass[index].thisRay.origin = firstRay.origin;
			firstPass[index].thisRay.direction = firstRay.direction;
			firstPass[index].dead = false;
			firstPass[index].pixID = index;
			compactIn[index] = 1;

			//glm::vec3 surfColor = materials[matID].color;
			////output final color
			//colors[index] += materials[matID].color;	
		}
		else{
			firstPass[index] = rayBounce();
			firstPass[index].dead = true;
			firstPass[index].pixID = index;
			compactIn[index] = 0;

		}
	}

}

__global__ void rayParallelTrace(glm::vec2 resolution, float time, cameraData cam, int maxDepth, glm::vec3* colors, staticGeom* geoms, int numberOfGeoms, 
								material* materials, int numLights, int* lightID, int numRays, int* compactIn, int* compactOut, rayBounce* rayPass){

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	glm::vec3 intersection;
	glm::vec3 normal;
	glm::vec3 surfColor;

	//int currDepth = 1;

	//rayBounce currBounce = rayPass[10];
	//colors[currBounce.pixID] = glm::vec3(1,0,0);
	
	if(index < numRays){
		
		compactIn[index] = 0;
		compactOut[index] = 0;

		rayBounce currBounce = rayPass[index];

		//normal = currBounce.normal;
		//glm::vec3 testColor (abs(normal[0]), abs(normal[1]), abs(normal[2]));
		//colors[index] += testColor;
		//colors[index] += glm::vec3(1,0,0);
		if(!currBounce.dead)
			colors[currBounce.pixID] += glm::vec3(1, 0, 0);
		else
			colors[currBounce.pixID] += glm::vec3(0, 0, 1);

		//ray firstRay = rayPass[index].thisRay;
		//firstRay.direction = glm::normalize(firstRay.direction - 2.0f*glm::dot(firstRay.direction, normal)*normal);
		//offsect a little to prevent intersection
		//firstRay.origin = intersection + 0.0001f * firstRay.direction;
		
		//glm::vec3 finalColor(1,1,1);
		
		//do intersection test
		//int objID = -1;

		//float len = testGeomIntersection(geoms, numberOfGeoms, firstRay, intersection, normal, objID);

		////if no intersection, return
		//if(objID == -1){
		//	rayPass[index].dead = true;
		//	compactIn[index] = 1;
		//	return;	
		//}
		//
		//int matID = geoms[objID].materialid;
		//
		////save the first bounce information
		//if(materials[matID].hasReflective == 1){
		//	rayPass[index].intersectPt = intersection;
		//	rayPass[index].normal = normal;
		//	rayPass[index].matID = matID;
		//	rayPass[index].thisRay.origin = firstRay.origin;
		//	rayPass[index].thisRay.direction = firstRay.direction;
		//	rayPass[index].dead = false;
		//	rayPass[index].pixID = index;
		//	compactIn[index] = 1;

		//	surfColor = materials[matID].color;
		//	//output final color
		//	colors[index] += surfColor;		
		//}
		//else{
		//	rayPass[index].dead = true;
		//	rayPass[index].pixID = index;
		//	compactIn[index] = 1;
		//}
	}

}

//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors, staticGeom* geoms, int numberOfGeoms, 
							material* materials, int numLights, int* lightID, rayBounce* firstPass){

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	glm::vec3 intersection;
	glm::vec3 normal;
	glm::vec3 surfColor;

	int currDepth = 1;

	if((x<=resolution.x && y<=resolution.y)){

		ray firstRay = raycastFromCameraKernel(resolution, x, y, cam.position, cam.view, cam.up, cam.fov); 
		glm::vec3 finalColor(1,1,1);
		
		while(currDepth <= rayDepth){

#pragma region setup
			//DOF setup thing
			float focalLength = cam.focalLength;

			float aperture = cam.aperture;
		
			glm::vec3 focalPoint = firstRay.origin + focalLength * firstRay.direction;

			//jitter camera
			glm::vec3 jitterVal = 2.0f * aperture * generateRandomNumberFromThread(resolution, time, x, y);
			jitterVal -= glm::vec3(aperture);
			firstRay.origin += jitterVal;

			//find new direction
			firstRay.direction = glm::normalize(focalPoint - firstRay.origin);

			//antialias sample per pixel
			jitterVal = generateRandomNumberFromThread(resolution, time, x, y);
			jitterVal -= glm::vec3(0.5f, 0.5f, 0.5f);
			firstRay.direction += 0.0015f* jitterVal; 

			//firstPass[index] = firstRay;
#pragma endregion setup

			//do intersection test
			int objID = -1;

			float len = testGeomIntersection(geoms, numberOfGeoms, firstRay, intersection, normal, objID);

			//if no intersection, return
			if(objID == -1){
				finalColor *= 0.0f;
				break;	
			}
		
			int matID = geoms[objID].materialid;
			surfColor = materials[matID].color;

			//check if you intersected with light, if so, just return light color
			if(materials[matID].emittance > 0){
				finalColor *= surfColor;
				break;
			}


	#pragma region lightAndShadow
			glm::vec3 diffuse(0,0,0);
			glm::vec3 phong(0,0,0);

			//do light and shadow computation
			for(int i = 0; i < numLights; ++i){

				int lightGeomID = lightID[i];
				glm::vec3 lightPos;
				glm::vec3 lightColor = materials[geoms[lightGeomID].materialid].color;

				//find a random point on the light
				if(geoms[lightGeomID].type == CUBE){
					lightPos = getRandomPointOnCube(geoms[lightGeomID], time);		//CHANGE TO TIME!
				}
				else if(geoms[lightGeomID].type == SPHERE){
					lightPos = getRandomPointOnSphere(geoms[lightGeomID], time);	//CHANGE TO TIME!
				}

				//find vector from intersection to point on light
				glm::vec3 L = lightPos - intersection;
				float distToLight = glm::length(L);
				L = glm::normalize(L);

				//check if in shadow
				objID = -1;
				ray shadowFeeler; 
				shadowFeeler.direction = L;
				shadowFeeler.origin = intersection + 0.0001f*L;		//offset origin a little bit so it doesn't self intersect
			
				glm::vec3 shadowNormal; glm::vec3 shadowIntersection;
				len = testGeomIntersection(geoms, numberOfGeoms, shadowFeeler, shadowIntersection, shadowNormal, objID);

				//if intersection occured and intersection is in between the intersection point and the light position
				if(objID != -1 && len < distToLight){
				
					if(materials[geoms[objID].materialid].emittance == 0){		//only cast shadow if we intersected with object that is not a light
						//color is ambient color
						finalColor = glm::vec3(0,0,0);
						continue;
					}
				}

				//do diffuse calculation
				diffuse += glm::clamp(glm::dot(L, normal), 0.0f, 1.0f) * surfColor * lightColor;
			
				//clamp diffuse to surface color
				diffuse.x = clamp(diffuse.x, 0.0f, surfColor.x);
				diffuse.y = clamp(diffuse.y, 0.0f, surfColor.y);
				diffuse.z = clamp(diffuse.z, 0.0f, surfColor.z);

				//specular
				if(materials[matID].specularExponent != 0){
					glm::vec3 R = glm::normalize( -L - 2.0f*glm::dot(-L, normal) *normal);
					glm::vec3 V = -firstRay.direction;			//already normalized
			
					phong += materials[matID].specularColor * 
							pow(glm::clamp(glm::dot(R, V), 0.0f, 1.0f), materials[matID].specularExponent) * lightColor;
					//phong *= 0.5f;
					//diffuse *= 0.9f;
				}

			}
	#pragma endregion lightAndShadow

			//check for reflection
			if(materials[matID].hasReflective == 1){
				//reflect
				firstRay.direction = glm::normalize(firstRay.direction - 2.0f*glm::dot(firstRay.direction, normal)*normal);
				
				//offsect a little to prevent intersection
				firstRay.origin = intersection + 0.0001f * firstRay.direction;
				currDepth++;
				
				finalColor *= glm::clamp(surfColor + phong, 0.0f, 1.0f);
			}
			else{
				finalColor *= glm::clamp(diffuse + phong, 0.0f, 1.0f);
				break;
			}

		}
	
		//output final color
		colors[index] += finalColor;
	}

}

__global__ void resetCompactVals(int* compactA, int* compactB, int imageSize){
	
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(index < imageSize)
		compactA[index] = compactB[index] = 0;
}

//allocate memory on cuda
void cudaAllocMemory(glm::vec2 resolution){

	int size = (int)resolution.x*resolution.y;
	
	//std::cout<<"allocate "<<std::endl;
	//cache the first bounce since they're the same for each iteration
	cudaFirstPass = NULL;
	hipMalloc((void**)&cudaFirstPass, size*sizeof(rayBounce));

	cudaRayPool = NULL;
	hipMalloc((void**)&cudaRayPool, size*sizeof(rayBounce));

	cudaTempRayPool = NULL;
	hipMalloc((void**)&cudaTempRayPool, size*sizeof(rayBounce));

	cudaCompactA= NULL;
	hipMalloc((void**)&cudaCompactA, size*sizeof(int));

	cudaCompactB= NULL;
	hipMalloc((void**)&cudaCompactB, size*sizeof(int));

}

void cudaFreeMemory(){
	//std::cout<<"free memory "<<std::endl;
	hipFree( cudaFirstPass);
	hipFree( cudaRayPool);
	hipFree( cudaTempRayPool);
	hipFree( cudaCompactA);
	hipFree( cudaCompactB);
}

//TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms, bool& clear){
  
  int traceDepth = 1; //determines how many bounces the raytracer traces

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);			//each block has 8 * 8 threads
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  //send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  std::vector<int> lightVec;

  //package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
  
	//store which objects are lights
	if(materials[geoms[i].materialid].emittance > 0)
		lightVec.push_back(i);
  }
  
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);

  //copy materials to memory
  material* cudaMaterials = NULL;
  hipMalloc((void**)&cudaMaterials, numberOfMaterials*sizeof(material));
  hipMemcpy( cudaMaterials, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

  //copy light ID to memeory
  int numLights = lightVec.size();
  int* lightID = new int[numLights];
  for(int i = 0; i <numLights; ++i)
	  lightID[i] = lightVec[i];
  
  int* cudaLights = NULL;
  hipMalloc((void**)&cudaLights, numLights*sizeof(int));
  hipMemcpy( cudaLights, lightID, numLights*sizeof(int), hipMemcpyHostToDevice);
  
  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;
  cam.focalLength = renderCam->focalLengths[frame];
  cam.aperture = renderCam->apertures[frame];

  int imageSize = (int)renderCam->resolution.x * (int)renderCam->resolution.y;
  int numRays = imageSize;

  //clear image if camera has been moved
  if(clear){
	  clearImage<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, cudaimage); 
	  clear = false;
  }
  else{
	  //first pass, get rays for first bounce
	  //if(iterations == 1) {
		  createRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, cam, cudageoms, numberOfGeoms, cudaMaterials, 
															numLights, cudaLights, cudaFirstPass, traceDepth, cudaCompactA, numRays, cudaimage);
	  //}

	  dim3 threadsPerBlockRayPool(tileSize*tileSize);			//each block has 64 * 1 threads
	  dim3 fullBlocksPerGridRayPool;

	  hipMemcpy(cudaTempRayPool, cudaFirstPass, imageSize*sizeof(rayBounce), hipMemcpyDeviceToDevice);		//copy new rays to the ray pool	  
	  hipMemcpy(cudaRayPool, cudaFirstPass, imageSize*sizeof(rayBounce), hipMemcpyDeviceToDevice);

	  for(int depthCount = 1; depthCount <= traceDepth; ++depthCount){  
		  
		  cudaStreamCompaction(fullBlocksPerGridRayPool, threadsPerBlockRayPool, tileSize, imageSize, traceDepth, numRays, depthCount);

		  //reset compaction matrices for next iteration
		  dim3 resetBlocksPerGrid((int)ceil(imageSize/float(tileSize)/float(tileSize)));
		  dim3 resetThreadsPerBlock(tileSize * tileSize);
		  //resetCompactVals<<<resetBlocksPerGrid, threadsPerBlockRayPool>>>(cudaCompactA, cudaCompactB, imageSize);

		  //run raytrace in parallel
		  rayParallelTrace<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms, 
															cudaMaterials, numLights, cudaLights, numRays, cudaCompactA, cudaCompactB, cudaRayPool);

		  hipMemcpy(cudaTempRayPool, cudaRayPool, imageSize*sizeof(rayBounce), hipMemcpyDeviceToDevice);
	 	  checkCUDAError("building raypool failed!");
	  }
  }

  //reset compaction matrices for next iteration
  dim3 resetBlocksPerGrid((int)ceil(imageSize/float(tileSize)/float(tileSize)));
  dim3 resetThreadsPerBlock(tileSize * tileSize);
  resetCompactVals<<<resetBlocksPerGrid, resetThreadsPerBlock>>>(cudaCompactA, cudaCompactB, imageSize);

  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage, (float)iterations);

  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, imageSize*sizeof(glm::vec3), hipMemcpyDeviceToHost);


  //free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  hipFree( cudaMaterials);
  hipFree( cudaLights);
  delete geomList;
  delete lightID;

  // make certain the kernel has completed
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}

void cudaStreamCompaction(dim3& fullBlocksPerGridRayPool, dim3 threadsPerBlockRayPool, int tileSize, int imageSize, int traceDepth, int& numRays, int currTraceDepth){

	int compactDepth = (int)ceil(log((float)imageSize) / log(2.0f));
	int compactStart = 0;

	fullBlocksPerGridRayPool = ((int)ceil(imageSize/float(tileSize)/float(tileSize)));

	for(int d = 1; d <= compactDepth; ++d){
		compactStart = pow(2.0f, d-1);
		//swap buffers every iteration
		if(d % 2 == 1){
			streamCompact<<<fullBlocksPerGridRayPool, threadsPerBlockRayPool>>>(numRays, cudaCompactA, cudaCompactB, traceDepth, compactStart);
			hipDeviceSynchronize();
			//std::cout<<testNum[0]<<std::endl;
		}
		else{
			streamCompact<<<fullBlocksPerGridRayPool, threadsPerBlockRayPool>>>(numRays, cudaCompactB, cudaCompactA, traceDepth, compactStart);
			hipDeviceSynchronize();
			//std::cout<<testNum[0]<<std::endl;
		}
		checkCUDAError("compact failed!");
	}

	int* newNumRays = new int[640000];

	if(compactStart %2 == 1){	
		shiftRight<<<fullBlocksPerGridRayPool, threadsPerBlockRayPool>>>(cudaCompactB, cudaCompactA, numRays);
		buildRayPool<<<fullBlocksPerGridRayPool, threadsPerBlockRayPool>>>(cudaCompactA, cudaTempRayPool, cudaRayPool, numRays);
		
		//hipMemcpy(newNumRays, cudaCompactB, imageSize*sizeof(int), hipMemcpyDeviceToHost);
		//numRays= newNumRays[imageSize-1];
		
		//find how many blocks you need now that you've killed rays
		//fullBlocksPerGridRayPool = ((int)ceil(numRays/float(tileSize)/float(tileSize)));

		//std::cout<<numRays<<std::endl;
	}
	else{

		//hipMemcpy(newNumRays, cudaCompactA, imageSize*sizeof(int), hipMemcpyDeviceToHost);
		//numRays= newNumRays[imageSize-1];
			
		//std::cout<<numRays<<std::endl;
			
		//fullBlocksPerGridRayPool = ((int)ceil(numRays/float(tileSize)/float(tileSize)));

		shiftRight<<<fullBlocksPerGridRayPool, threadsPerBlockRayPool>>>(cudaCompactA, cudaCompactB, numRays);
		buildRayPool<<<fullBlocksPerGridRayPool, threadsPerBlockRayPool>>>(cudaCompactB, cudaTempRayPool, cudaRayPool, numRays);

	}

	//int count = 0;
	//for(int i =0 ; i< numRays; ++i)
	//	if(newNumRays[i] > 1){
	//		count++;
	//		//std::cout<<newNumRays[i]<<" ";
	//	}

	//std::cout<<count<<std::endl;

	delete [] newNumRays;
		
	checkCUDAError("building raypool failed!");

}

