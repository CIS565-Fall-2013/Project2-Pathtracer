#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <ctime>
#include <random>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

const glm::vec3 bgColour = glm::vec3 (0.55, 0.25, 0);

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
	std::cin.get ();
    exit(EXIT_FAILURE); 
  }
} 

//Sets up the projection half vectors.
void	setupProjection (projectionInfo &ProjectionParams, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov)
{
	//Set up the projection variables:
	float	degToRad = 3.1415926 / 180.0;
	float	radToDeg = 1.0 / degToRad;

	ProjectionParams.centreProj = eye+view;
	glm::vec3	eyeToProjCentre = ProjectionParams.centreProj - eye;
	glm::vec3	A = glm::cross (ProjectionParams.centreProj, up);
	glm::vec3	B = glm::cross (A, ProjectionParams.centreProj);
	float		lenEyeToProjCentre = glm::length (eyeToProjCentre);
	
	ProjectionParams.halfVecH = glm::normalize (A) * lenEyeToProjCentre * (float)tan ((fov.x*degToRad));
	ProjectionParams.halfVecV = glm::normalize (B) * lenEyeToProjCentre * (float)tan ((fov.y*degToRad));
}

// Reflects the incidentRay around the normal.
__host__ __device__ glm::vec3 reflectRay (glm::vec3 incidentRay, glm::vec3 normal)
{
	glm::vec3 reflectedRay = incidentRay - (2.0f*glm::dot (incidentRay, normal))*normal;
	return reflectedRay;
}

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov, glm::vec3 centreProj,
													glm::vec3	halfVecH, glm::vec3 halfVecV)
{
  ray r;
  r.origin = eye;
  r.direction = glm::vec3(0,0,-1);

  float normDeviceX = (float)x / (resolution.x-1);
  float normDeviceY = 1 - ((float)y / (resolution.y-1));

  glm::vec3 P = centreProj + (2*normDeviceX - 1)*halfVecH + (2*normDeviceY - 1)*halfVecV;
  r.direction = glm::normalize (P - r.origin);

  return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image, int nLights){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){
	  image [index] /= nLights;
      glm::vec3 color;
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

__device__ interceptInfo getIntercept (staticGeom * geoms, sceneInfo objectCountInfo, ray castRay, material* textureArray)
{
	glm::vec3 intrPoint = glm::vec3 (0, 0, 0);
	glm::vec3 intrNormal = glm::vec3 (0, 0, 0);
	glm::vec2 UVcoords = glm::vec2 (0, 0);

	float interceptValue = -32767;

	material newMaterial;
	newMaterial.color = glm::vec3 (0,0,0);
	newMaterial.specularExponent = 1.0;
	newMaterial.hasReflective = 0.0;
	newMaterial.hasRefractive = 0.0;

	interceptInfo theRightIntercept;					// Stores the lowest intercept.
	theRightIntercept.interceptVal = interceptValue;	// Initially, it is empty/invalid
	theRightIntercept.intrNormal = intrNormal;			// Intially, Normal - 0,0,0
	theRightIntercept.intrMaterial = newMaterial;

	float min = 1e6;
	// Two different loops to intersect ray with cubes and spheres.
	for (int i = 0; i < objectCountInfo.nCubes; ++i)
	{
		staticGeom currentGeom = geoms [i];

		interceptValue = boxIntersectionTest(currentGeom, castRay, intrPoint, intrNormal, UVcoords);
		if (interceptValue > 0)
		{
			if (interceptValue < min)
			{
				min = interceptValue;

				theRightIntercept.interceptVal = min;
				theRightIntercept.intrNormal = intrNormal;
				theRightIntercept.intrMaterial = textureArray [currentGeom.materialid];
				theRightIntercept.UV = UVcoords;
			}
		}
	}

	for (int i = objectCountInfo.nCubes; i <= (objectCountInfo.nCubes+objectCountInfo.nSpheres); ++i)
	{
		staticGeom currentGeom = geoms [i];

		interceptValue = sphereIntersectionTest(currentGeom, castRay, intrPoint, intrNormal);
		if (interceptValue > 0)
		{
			if (interceptValue < min)
			{
				min = interceptValue;

				theRightIntercept.interceptVal = min;
				theRightIntercept.intrNormal = intrNormal;
				theRightIntercept.intrMaterial = textureArray [currentGeom.materialid];
			}
		}
	}

	return theRightIntercept;
}

__device__ unsigned long getIndex (int x, int y, int MaxWidth)
{	return (unsigned long) y*MaxWidth + x ;	}

__host__ __device__ bool isApproximate (float valToBeCompared, float valToBeCheckedAgainst) 
{ if ((valToBeCompared >= valToBeCheckedAgainst-0.001) && (valToBeCompared <= valToBeCheckedAgainst+0.001)) return true;	return false; }

__device__ glm::vec3 getColour (material Material, glm::vec2 UVcoords)
{
	if (Material.hasTexture)
	{	
		unsigned long texelXY, texelXPlusOneY, texelXYPlusOne, texelXPlusOneYPlusOne;
		float xInterp = (Material.Texture.texelWidth * UVcoords.x) - floor (Material.Texture.texelWidth * UVcoords.x);
		float yInterp = (Material.Texture.texelHeight * UVcoords.y) - floor (Material.Texture.texelHeight * UVcoords.y);

		texelXY = getIndex ((int)floor (Material.Texture.texelWidth * UVcoords.x), (int)floor (Material.Texture.texelHeight * UVcoords.y), Material.Texture.texelWidth);
		texelXPlusOneY = getIndex ((int)ceil (Material.Texture.texelWidth * UVcoords.x), (int)floor (Material.Texture.texelHeight * UVcoords.y), Material.Texture.texelWidth);
		texelXYPlusOne = getIndex ((int)floor (Material.Texture.texelWidth * UVcoords.x), (int)ceil (Material.Texture.texelHeight * UVcoords.y), Material.Texture.texelWidth);
		texelXPlusOneYPlusOne = getIndex ((int)ceil (Material.Texture.texelWidth * UVcoords.x), (int)ceil (Material.Texture.texelHeight * UVcoords.y), Material.Texture.texelWidth);

		glm::vec3 xInterpedColour1, xInterpedColour2, finalColour;
		xInterpedColour1 = xInterp * Material.Texture.texels [texelXPlusOneY] + (1-xInterp)* Material.Texture.texels [texelXY];
		xInterpedColour2 = xInterp * Material.Texture.texels [texelXPlusOneYPlusOne] + (1-xInterp)* Material.Texture.texels [texelXYPlusOne];
		finalColour = yInterp * xInterpedColour2 + (1-yInterp) * xInterpedColour1;

		return finalColour;
	}
	else
		return Material.color;
}

__device__ glm::vec3 calcShade (interceptInfo theRightIntercept, glm::vec3 lightVec, glm::vec3 eye, ray castRay, material* textureArray, float ks, float kd, glm::vec3 lightCol, float emittance)
{
	glm::vec3 shadedColour = glm::vec3 (0,0,0);
	if (theRightIntercept.interceptVal > 0)
	{
//		shadedColour = theRightIntercept.intrMaterial.color;
		// Diffuse shading
		glm::vec3 intrPoint = castRay.origin + theRightIntercept.interceptVal*castRay.direction;	// The intersection point.
		glm::vec3 intrNormal = glm::normalize (eye - intrPoint); // intrNormal is the view vector.
		float interceptValue = max (glm::dot (theRightIntercept.intrNormal, lightVec), (float)0); // Diffuse Lighting is given by (N.L); N being normal at intersection pt and L being light vector.
		intrPoint = (getColour (theRightIntercept.intrMaterial, theRightIntercept.UV) * kd * interceptValue);			// Reuse intrPoint to store partial product (kdId) of the diffuse shading computation.
		shadedColour += multiplyVV (lightCol*emittance, intrPoint);		// shadedColour will have diffuse shaded colour. 
		// Quick and Dirty fix for lights.
		if ((theRightIntercept.intrMaterial.emittance > 0) && (interceptValue > 0))
			shadedColour = glm::vec3 (1,1,1);
		
		// Specular shading
		lightVec = glm::normalize (reflectRay (-lightVec, theRightIntercept.intrNormal)); // Reuse lightVec for storing the reflection of light vector around the normal.
		interceptValue = max (glm::dot (lightVec, intrNormal), (float)0);				// Reuse interceptValue for computing dot pdt of specular.
		shadedColour += (lightCol * ks * pow (interceptValue, theRightIntercept.intrMaterial.specularExponent));
	}

	return	shadedColour;
}

//TODO: Done!
//Core raytracer kernel
__global__ void raytraceRay (float time, cameraData cam, int rayDepth, glm::vec3* colors, staticGeom* geoms, 
							 material* textureArray, renderInfo * RenderParams, sceneInfo objectCountInfo, 
							 bool *primaryArrayOnDevice, ray *rayPoolOnDevice, int rayPoolLength, glm::vec3 lightPosition)
{
  __shared__ staticGeom light;
  __shared__ renderInfo RenderParamsOnBlock;
  __shared__ float ks;
  __shared__ float kd;
  __shared__ glm::vec3 lightPos;
  __shared__ glm::vec3 lightCol;
  __shared__ float nLights;
  __shared__ int sqrtLights;
  __shared__ float stepSize;
  __shared__ float lightEmittance;

  extern __shared__ glm::vec3 arrayPool [];
  __shared__ glm::vec3 *colourBlock; 
  __shared__ bool *primArrayBlock;
  __shared__ ray *rayPoolBlock;

  if ((threadIdx.x == 0) && (threadIdx.y == 0))
  {
	  RenderParamsOnBlock = *RenderParams;
	  ks = RenderParams->ks;
	  kd = RenderParams->kd;
	  nLights = RenderParams->nLights;
	  sqrtLights = RenderParams->sqrtLights;
	  stepSize = RenderParams->lightStepSize;
	  light = geoms [0];
	  lightPos = lightPosition;
	  lightCol = RenderParams->lightCol;
	  lightEmittance = textureArray [light.materialid].emittance;

	  colourBlock = arrayPool;
	  primArrayBlock = (bool *) &colourBlock [blockDim.x * blockDim.y];
	  rayPoolBlock = (ray *) &primArrayBlock [blockDim.x * blockDim.y];
	  for (int i = 0; i < blockDim.y; i ++)
		  for (int j = 0; j < blockDim.x; j ++)
		  {
			  // We have a 1-D array of blocks in the grid. From a thread's perspective, it is a 2-D array.
			  // Ray pool is a massive 1-D array, so we need to compute the index of the element of ray pool
			  // that each thread will handle.
			  int index = (blockIdx.x * blockDim.x) + j +			// X-part: straightforward
						  (i * (int)(blockDim.x * ceil ((float)rayPoolLength / (float)(blockDim.x*blockDim.y))));  // Y-part: as below:
			  // No. of blocks in the grid = ceil (rayPoolLength / (blockDim.x*blockDim.y))
			  // Multiplying that with the no. threads in a block gives the no. of threads in a single row of grid.
			  // Multiplying that with row number (threadIdx.y) and adding the x offset (X-part) gives the index.
			  int index2 = i*blockDim.x + j;
			  primArrayBlock [index2] = primaryArrayOnDevice [index];
			  rayPoolBlock [index2] = rayPoolOnDevice [index];

			  // We recompute the index for the colour array since it represents a frame
			  // and each index represents a pixel. If we don't, stream compaction would 
			  // mess things up.
			  index = rayPoolBlock [index2].y*cam.resolution.x + rayPoolBlock [index2].x;
 			  colourBlock [index2] = colors [index];
			  // colourBlock [index2] therefore represents colour computed by ray through the pixel (x,y)
		  }
  }
  __syncthreads ();


  int threadID = (blockIdx.x * blockDim.x) + threadIdx.x +			
				 (threadIdx.y * (int)(blockDim.x * ceil ((float)rayPoolLength / (float)(blockDim.x*blockDim.y))));
  
  glm::vec3 shadedColour = glm::vec3 (0);
  if (threadID < rayPoolLength)
  {
	threadID = threadIdx.y*blockDim.x + threadIdx.x;
	ray currentRay = rayPoolBlock [threadID];
	interceptInfo theRightIntercept = getIntercept (geoms, objectCountInfo, currentRay, textureArray);
	glm::vec3 lightVec; 
		
	lightVec = glm::normalize (lightPosition - (currentRay.origin + (currentRay.direction*theRightIntercept.interceptVal)));
	shadedColour += calcShade (theRightIntercept, lightVec, cam.position, currentRay, textureArray, ks, kd, lightCol, lightEmittance);

	if ((theRightIntercept.intrMaterial.emittance > 0) || (theRightIntercept.interceptVal < 0))
		primArrayBlock [threadID] = false;	// Ray did not hit anything or it hit light, so kill it.
	else
		calculateBSDF  (currentRay, 
						currentRay.origin + currentRay.direction * theRightIntercept.interceptVal, 
						theRightIntercept.intrNormal, glm::vec3 (0), AbsorptionAndScatteringProperties (), 
						time, theRightIntercept.intrMaterial.color, glm::vec3 (0), theRightIntercept.intrMaterial);

	rayPoolBlock [threadID] = currentRay;
	
	if (glm::length (colourBlock [threadID]) > 0)
		colourBlock [threadID] *= shadedColour;			// Add computed shade to shadedColour.
	else
		colourBlock [threadID] = shadedColour;
  }

  __syncthreads ();

  if ((threadIdx.x ==0) && threadIdx.y == 0)
  {
	  for (int i = 0; i < blockDim.y; i ++)
		  for (int j = 0; j < blockDim.x; j ++)
		  {
			  // Calculate the index.
			  int index = (blockIdx.x * blockDim.x) + j +												// X-part
						  (i * (int)(blockDim.x * ceil ((float)rayPoolLength / (float)(blockDim.x*blockDim.y))));		// Y-part

			  threadID = i*blockDim.x + j;
			  primaryArrayOnDevice [index] = primArrayBlock [threadID];
			  rayPoolOnDevice [index] = rayPoolBlock [threadID];

			  // Recalculate the index for colour array.
			  index = rayPoolBlock [threadID].y*cam.resolution.x + rayPoolBlock [threadID].x;
			  colors [index] = colourBlock [threadID];
		  }
  }
}

__device__ bool isShadowRayBlocked (ray r, glm::vec3 lightPos, staticGeom *geomsList, sceneInfo objectCountInfo)
{
	float min = 1e6, interceptValue;
	glm::vec3 intrPoint, intrNormal;
	glm::vec2 UVcoords = glm::vec2 (0, 0);
	for (int i = 0; i < objectCountInfo.nCubes; ++i)
	{
		staticGeom currentGeom = geomsList [i];
		interceptValue = boxIntersectionTest(currentGeom, r, intrPoint, intrNormal, UVcoords);
		if (interceptValue > 0)
		{
			if (interceptValue < min)
				min = interceptValue;
		}
	}

	for (int i = objectCountInfo.nCubes; i <= (objectCountInfo.nCubes+objectCountInfo.nSpheres); ++i)
	{
		staticGeom currentGeom = geomsList [i];
		interceptValue = sphereIntersectionTest(currentGeom, r, intrPoint, intrNormal);
		if (interceptValue > 0)
		{
			if (interceptValue < min)
				min = interceptValue;
		}
	}

	if (glm::length (lightPos - r.origin) > (min+0.1))
		return true;
	return false;
}

__global__ void		accumulateIterationColour (glm::vec3* accumulator, glm::vec3* iterationColour, glm::vec2 resolution)
{
	int index = (blockDim.y*blockIdx.y + threadIdx.y) * resolution.x + 
				(blockDim.x*blockIdx.x + threadIdx.x);
	accumulator [index] += iterationColour [index];
}

// If errorCode is not hipSuccess, kills the program.
void onDeviceErrorExit (hipError_t errorCode, glm::vec3 *cudaimage, staticGeom *cudageoms, material * materialColours, int numberOfMaterials)
{
  if (errorCode != hipSuccess)
  {
	  std::cout << "\nError while trying to send texture data to the GPU!";
	  std::cin.get ();

	  if (cudaimage)
		hipFree( cudaimage );
	  if (cudageoms)
		hipFree( cudageoms );
	  if (materialColours)
	  {
		   /*for (int i = 0; i < numberOfMaterials; i ++)
		   {
			   if (materialColours [i].hasTexture)
				hipFree (materialColours[i].Texture.texels);

			   if (materialColours [i].hasNormalMap)
				hipFree (materialColours[i].NormalMap.texels);
		   }*/
		  hipFree (materialColours);
	  }

	  cudaimage = NULL;
	  cudageoms = NULL;
	  materialColours = NULL;

	  exit (EXIT_FAILURE);
  }
}

//TODO: Done!
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){
  
  int traceDepth = 1; //determines how many bounces the raytracer traces
  projectionInfo	ProjectionParams;
  float degToRad = 3.1415926 / 180.0;
	
  // Set up projection.
	ProjectionParams.centreProj = renderCam->positions [frame]+renderCam->views [frame];
	glm::vec3	eyeToProjCentre = ProjectionParams.centreProj - renderCam->positions [frame];
	glm::vec3	A = glm::cross (eyeToProjCentre, renderCam->ups [frame]);
	glm::vec3	B = glm::cross (A, eyeToProjCentre);
	float		lenEyeToProjCentre = glm::length (eyeToProjCentre);
	
	ProjectionParams.halfVecH = glm::normalize (A) * lenEyeToProjCentre * (float)tan ((renderCam->fov.x*degToRad) / 2.0);
	ProjectionParams.halfVecV = glm::normalize (B) * lenEyeToProjCentre * (float)tan ((renderCam->fov.y*degToRad) / 2.0);

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)));
  
  //send image to GPU
  glm::vec3* cudaFinalImage = NULL;
  hipMalloc((void**)&cudaFinalImage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaFinalImage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);

  // package geometry to be sent to GPU global memory
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  sceneInfo		primCounts;
  
  // Reorder geometry so that light is the first item in geomList,
  // followed by cubes and then spheres. Doing so reduces divergence.
  int count = 1;	int lightIndex = 0;
  bool lightSet = false;
  for(int i=0; i<numberOfGeoms; i++)
  {
	  if ((geoms [i].materialid == 8) && !lightSet)
	  {
		staticGeom newStaticGeom;
		newStaticGeom.type = geoms[i].type;
		newStaticGeom.materialid = geoms[i].materialid;
		newStaticGeom.translation = geoms[i].translations[frame];
		newStaticGeom.rotation = geoms[i].rotations[frame];
		newStaticGeom.scale = geoms[i].scales[frame];
		newStaticGeom.transform = geoms[i].transforms[frame];
		newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
		geomList[0] = newStaticGeom;
		
		lightSet = true;
		lightIndex = i;
	  }

	  else if (geoms [i].type == CUBE)
	  {
		staticGeom newStaticGeom;
		newStaticGeom.type = geoms[i].type;
		newStaticGeom.materialid = geoms[i].materialid;
		newStaticGeom.translation = geoms[i].translations[frame];
		newStaticGeom.rotation = geoms[i].rotations[frame];
		newStaticGeom.scale = geoms[i].scales[frame];
		newStaticGeom.transform = geoms[i].transforms[frame];
		newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
		geomList[count] = newStaticGeom;
		count ++;
	  }
  }

  if (!lightSet)
  {
	  geomList [0] = geomList [count-1];
	  count --;
  }
  // Lights may only be cubes.
  primCounts.nCubes = count;
  
  for(int i=0; i<numberOfGeoms; i++)
  {
	  if (geoms [i].type == SPHERE)
	  {
		staticGeom newStaticGeom;
		newStaticGeom.type = geoms[i].type;
		newStaticGeom.materialid = geoms[i].materialid;
		newStaticGeom.translation = geoms[i].translations[frame];
		newStaticGeom.rotation = geoms[i].rotations[frame];
		newStaticGeom.scale = geoms[i].scales[frame];
		newStaticGeom.transform = geoms[i].transforms[frame];
		newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
		geomList[count] = newStaticGeom;
		count ++;
	  }
  }

  primCounts.nSpheres = count - primCounts.nCubes;
  primCounts.nMeshes = 0;

  // Allocate memory. We'll copy it later (because we're moving objects around for Motion blur).
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom)); 
  

  // Copy materials to GPU global memory:
  material		*materialColours = NULL;
  glm::vec3		*colourArray = NULL;

  // Guard against shallow copying here.. Materials has a pointer pointing to Texture data.
  int sizeOfMaterialsArr = numberOfMaterials * (sizeof (material));
  hipError_t returnCode1 = hipMalloc((void**)&materialColours, numberOfMaterials*sizeof(material));
  onDeviceErrorExit (returnCode1, cudaFinalImage, cudageoms, materialColours, numberOfMaterials);
  hipMemcpy (materialColours, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

  // TODO: Texture mapping: Use index to a texture array.
  // Deep copying textures and normal maps:
  //glm::vec3 *texture = NULL;
  //glm::vec3 *norMap = NULL;
  //material *copyMaterial = new material [numberOfMaterials];	// SUCKS!
  //for (int i = 0; i < numberOfMaterials; i ++)
  //{
	 // copyMaterial [i] = materials [i];
	 // copyMaterial [i].Texture.texels = NULL;
	 // copyMaterial [i].NormalMap.texels = NULL;
	 // int noOfTexels = 0, noOfNMapTexels = 0;
	 // if (copyMaterial [i].hasTexture)
	 // {
		//  noOfTexels = materials [i].Texture.texelHeight * materials [i].Texture.texelWidth;
		//  hipError_t returnCode2 = hipMalloc ((void **)&texture, noOfTexels * sizeof (glm::vec3));
		//  onDeviceErrorExit (returnCode2, cudaimage, cudageoms, materialColours, numberOfMaterials);
		//  copyMaterial [i].Texture.texels = texture;
	 // }

	 // if (copyMaterial [i].hasNormalMap)
	 // {
		//  noOfNMapTexels = materials [i].NormalMap.texelHeight * materials [i].NormalMap.texelWidth;
		//  hipError_t returnCode2 = hipMalloc ((void **)&norMap, noOfNMapTexels * sizeof (glm::vec3));
		//  onDeviceErrorExit (returnCode2, cudaimage, cudageoms, materialColours, numberOfMaterials);
		//  copyMaterial [i].NormalMap.texels = norMap;
	 // }
  //}

  //hipMemcpy (materialColours, copyMaterial, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

  //for (int i = 0; i < numberOfMaterials; i ++)
  //{

	 // if (noOfTexels)
		//  hipMemcpy( curMaterialDevice->Texture.texels, materials [i].Texture.texels, noOfTexels*sizeof(glm::vec3), hipMemcpyHostToDevice);
	 // if (noOfNMapTexels)
		//  hipMemcpy (curMaterialDevice->NormalMap.texels, materials [i].NormalMap.texels, noOfNMapTexels*sizeof(glm::vec3), hipMemcpyHostToDevice);

  //}

  // Need to check whether the above method is correct.

  // Copy the render parameters like ks, kd values, the no. of times the area light is sampled, 
  // the position of the light samples w/r to the light's geometry and so on.
  renderInfo	RenderParams, *RenderParamsOnDevice = NULL;
  RenderParams.ks = 0.4;
  RenderParams.kd = 1 - RenderParams.ks;
  RenderParams.nLights = 64;
  RenderParams.sqrtLights = sqrt ((float)RenderParams.nLights);
  RenderParams.lightStepSize = 1.0/(RenderParams.sqrtLights-1);
  RenderParams.lightPos = glm::vec3 (-0.5, -0.6, -0.5);
  RenderParams.lightCol = materials [geoms [lightIndex].materialid].color;
  hipMalloc ((void **)&RenderParamsOnDevice, sizeof (renderInfo));
  hipMemcpy (RenderParamsOnDevice, &RenderParams, sizeof (renderInfo), hipMemcpyHostToDevice);

  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;

  time_t startTime = time (NULL);
  std::default_random_engine randomNumGen (hash (startTime));
  std::uniform_real_distribution<float> jitter ((float)0, (float)0.142);

  float movement = 1.0/48;
  int nBounces = 4;

  // For each point sampled in the area light, launch the raytraceRay Kernel which will compute the diffuse, specular, ambient
  // and shadow colours. It will also compute reflected colours for reflective surfaces.
  for (int i = 0; i < RenderParams.nLights; i ++)
  {
	  glm::vec3* cudaimage = NULL;
	  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
	  hipMemset (cudaimage, 0, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));


	  float zAdd = jitter (randomNumGen);
	  float xAdd = jitter (randomNumGen); 
	  glm::vec3 curLightSamplePos = glm::vec3 (RenderParams.lightPos.x + ((i%RenderParams.sqrtLights)*RenderParams.lightStepSize), 
												RenderParams.lightPos.y, 
												RenderParams.lightPos.z + ((i/RenderParams.sqrtLights)*RenderParams.lightStepSize));
	  
	  // Area light sampled in a jittered grid to reduce banding.
	  curLightSamplePos.z += zAdd;
	  curLightSamplePos.x += xAdd;
	  
	 // if (!(i%8))	// Supersampling at 8x!
	 // {
		//cam.position.y += zAdd*0.002;
		//cam.position.x += xAdd*0.002;
	 // }

	  if (!(i/32))	// Motion blur!
	  {
		  geomList [primCounts.nCubes].translation += glm::vec3 (movement, 0, 0);
		  glm::mat4 transform = utilityCore::buildTransformationMatrix(geomList [primCounts.nCubes].translation, 
																	   geomList [primCounts.nCubes].rotation, 
																	   geomList [primCounts.nCubes].scale);
		  geomList [primCounts.nCubes].transform = utilityCore::glmMat4ToCudaMat4(transform);
		  geomList [primCounts.nCubes].inverseTransform = utilityCore::glmMat4ToCudaMat4(glm::inverse(transform));
	  }
	  // Now copy the geometry list to the GPU global memory.
	  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);

	  glm::vec3 lightPos = multiplyMV (geomList [0].transform, glm::vec4 (curLightSamplePos, 1.0));
	  
	  // Create Ray Pool. 
	  int rayPoolLength = cam.resolution.x * cam.resolution.y;
	  ray *rayPool = new ray [rayPoolLength];
	  // Initialize ray pool with rays passing through every pixel in projection plane.
	  for (int i=0; i < cam.resolution.y; i ++)
		  for (int j = 0; j < cam.resolution.x; j ++)
		  {
			  rayPool [i * (int)cam.resolution.x + j] = raycastFromCameraKernel (cam.resolution, iterations, j, i, cam.position, 
													cam.view, cam.up, cam.fov, ProjectionParams.centreProj, 
													ProjectionParams.halfVecH, ProjectionParams.halfVecV);
			  rayPool [i * (int)cam.resolution.x + j].x = j;
			  rayPool [i * (int)cam.resolution.x + j].y = i;
		  }
	  // Send ray pool to device.
	  ray *rayPoolOnDevice = NULL;
	  hipMalloc ((void **)&rayPoolOnDevice, rayPoolLength * sizeof (ray));
	  hipMemcpy (rayPoolOnDevice, rayPool, rayPoolLength * sizeof (ray), hipMemcpyHostToDevice);

	  // Create primary and secondary arrays for stream compaction:
	  bool *primaryArray = new bool [rayPoolLength];
	  bool *primaryArrayOnDevice = NULL;
	  hipMalloc ((void **)&primaryArrayOnDevice, rayPoolLength * sizeof (bool));
	  hipMemset (primaryArrayOnDevice, true, rayPoolLength * sizeof (bool)); // (primaryArrayOnDevice, primaryArray, rayPoolLength * sizeof (bool), hipMemcpyHostToDevice);
	  
	  int *secondaryArray = new int [rayPoolLength];
	  memset (secondaryArray, 0, rayPoolLength * sizeof (int));

//	  int *secondaryArrayOnDevice = NULL;
//	  hipMalloc ((void **)&secondaryArrayOnDevice, rayPoolLength * sizeof (int));
//	  hipMemset (secondaryArrayOnDevice, 0, rayPoolLength * sizeof (int)); // (primaryArrayOnDevice, primaryArray, rayPoolLength * sizeof (bool), hipMemcpyHostToDevice);
	  

	  // Iterate until nBounces: launch kernel to trace each ray bounce.
	  for (int i = 0; i < nBounces; ++i)
	  {
		// kernel launches
		fullBlocksPerGrid = dim3 ((int)ceil(float(rayPoolLength)/(threadsPerBlock.x*threadsPerBlock.y))); 
		raytraceRay<<<fullBlocksPerGrid, threadsPerBlock, threadsPerBlock.x*threadsPerBlock.y*(sizeof(glm::vec3) + sizeof (bool) + sizeof(ray))>>>
			((float)iterations, cam, traceDepth, cudaimage, cudageoms, materialColours, RenderParamsOnDevice, 
			 primCounts, primaryArrayOnDevice, rayPoolOnDevice, rayPoolLength, lightPos);
		hipDeviceSynchronize(); // Wait for Kernel to finish, because we don't want a race condition between successive kernel launches.
		checkCUDAError("raytraceRay Kernel failed!");

		// Inefficient. Grossly inefficient. Need to look over and change as required.
		hipMemcpy (primaryArray, primaryArrayOnDevice, rayPoolLength * sizeof (bool), hipMemcpyDeviceToHost);
		hipMemcpy (rayPool, rayPoolOnDevice, rayPoolLength * sizeof (ray), hipMemcpyDeviceToHost);

		// Stream compaction:
		secondaryArray [0] = 0;
		for (int i = 1; i < rayPoolLength; ++ i)
			secondaryArray [i] = secondaryArray [i-1] + primaryArray [i-1];

		int count = 0;
		for (int i = 0; i < rayPoolLength; ++ i)
		{
			if (primaryArray [i])
			{
				rayPool [count] = rayPool [secondaryArray [i]];
				++ count;
			}
		}

		rayPoolLength = count;
		hipMemcpy (rayPoolOnDevice, rayPool, rayPoolLength * sizeof (ray), hipMemcpyHostToDevice);
		hipMemset (primaryArrayOnDevice, true, rayPoolLength * sizeof (bool));
	  }
	  fullBlocksPerGrid = dim3 ((int)ceil(float(cam.resolution.x)/threadsPerBlock.x), (int)ceil(float(cam.resolution.y)/threadsPerBlock.y));
	  // Validated:
	  accumulateIterationColour<<<fullBlocksPerGrid, threadsPerBlock>>>(cudaFinalImage, cudaimage, cam.resolution);
	  checkCUDAError("accumulateIterationColour Kernel failed!");

	  delete [] rayPool;
	  delete [] primaryArray;
	  delete [] secondaryArray;

	  hipFree (rayPoolOnDevice);
	  hipFree (primaryArrayOnDevice);
	  hipFree (cudaimage);

	  rayPoolOnDevice = NULL;
	  primaryArrayOnDevice = NULL;
	  cudaimage = NULL;

	  std::cout << "\rRendering.. " <<  ceil ((float)i/(RenderParams.nLights-1) * 100) << " percent complete.";
  }

  // Accumulate all the colours in the cudaFinalImage memory block on the GPU, and divide 
  // by the no. of light samples to get the final colour.
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaFinalImage, RenderParams.nLights);
  std::cout.precision (2);
  std::cout << "\nRendered in " << difftime (time (NULL), startTime) << " seconds. \n\n";
  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaFinalImage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

  //free up stuff, or else we'll leak memory like a madman
   if (cudaFinalImage)
		hipFree( cudaFinalImage );
   if (cudageoms)
		hipFree( cudageoms );
   if (materialColours)
   {
	   /*for (int i = 0; i < numberOfMaterials; i ++)
	   {
		   if (materialColours [i].hasTexture)
			hipFree (materialColours[i].Texture.texels);

		   if (materialColours [i].hasNormalMap)
			hipFree (materialColours[i].NormalMap.texels);
	   }*/
	   hipFree (materialColours);
   }

   cudaFinalImage = NULL;
   cudageoms = NULL;
   materialColours = NULL;

 // make certain the kernel has completed
  hipDeviceSynchronize();
  
  delete [] geomList;

  checkCUDAError("Kernel failed!");
  
//  std::cin.get ();
}
