#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <vector>
#include <time.h>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"

using glm::vec3;
using glm::cross;
using glm::length;
using glm::dot;
using glm::normalize;


#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
//Thrust doc: https://developer.nvidia.com/thrust
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

__host__ __device__ glm::vec3 generateRandomOffsetFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(-0.5,0.5);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}


__host__ __device__ void applyDepthOfField(ray &r, glm::vec2 resolution, float time, int x, int y)
{
	float dofDist = 12.0f; // adjust this to change the focal length
	vec3 offset = generateRandomOffsetFromThread(resolution, time, x, y);
	vec3 focusPoint = r.origin + dofDist * r.direction;
	vec3 jitteredOrigin = r.origin + offset;
	r.direction = glm::normalize(focusPoint - jitteredOrigin);
	r.origin = jitteredOrigin;
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, float x, float y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov)
{
	ray r;
	float width = resolution.x;
	float height = resolution.y;
	vec3 M = eye + view;
	vec3 A = cross(view, up);
	vec3 B = cross(A, view);
	vec3 H = (A * length(view) * tanf(fov.x * ((float)PI/180.0f))) / length(A);
	vec3 V = -(B * length(view) * tanf(fov.y * ((float)PI/180.0f))) / length(B); // LOOK: Multiplied by negative to flip the image
	vec3 P = M + ((2.0f*x)/(width-1)-1)*H + ((2.0f*y)/(height-1)-1)*V;
	vec3 D = P - eye;
	vec3 DN = glm::normalize(D);

	r.origin = P;
	r.direction = DN;

	if (DEPTH_OF_FIELD_SWITCH)
		applyDepthOfField(r, resolution, time, x, y);

	return r;
}

//Function that does the initial raycast from the camera with small jittered offset
__host__ __device__ ray jitteredRaycastFromCameraKernel(glm::vec2 resolution, float time, float x, float y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov)
{
	ray r;
	float width = resolution.x;
	float height = resolution.y;
	vec3 M = eye + view;
	vec3 A = cross(view, up);
	vec3 B = cross(A, view);
	vec3 H = (A * length(view) * tanf(fov.x * ((float)PI/180.0f))) / length(A);
	vec3 V = -(B * length(view) * tanf(fov.y * ((float)PI/180.0f))) / length(B); // LOOK: Multiplied by negative to flip the image

	vec3 offset = generateRandomOffsetFromThread(resolution, time, x, y);

	// offset the point by a small random number ranging from [-0.5,0.5] for anti-aliasing
	vec3 P = M + ((2.0f*(offset.x+x))/(width-1)-1)*H + ((2.0f*(offset.y+y))/(height-1)-1)*V;
	vec3 D = P - eye;
	vec3 DN = glm::normalize(D);

	r.origin = P;
	r.direction = DN;

	if (DEPTH_OF_FIELD_SWITCH)
		applyDepthOfField(r, resolution, time, x, y);

	return r;
}


//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, float iteration, glm::vec2 resolution, glm::vec3* image, glm::vec3* imageAccumd){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y)
  {
      glm::vec3 color;
	  
	  //image[index] = glm::clamp(image[index], vec3(0,0,0), vec3(1,1,1)); // Note: Commenting this out makes the image a lot brighter.

	  imageAccumd[index] = (imageAccumd[index] * (iteration - 1) + image[index]) / iteration;

	  imageAccumd[index] = glm::clamp(imageAccumd[index], vec3(0,0,0), vec3(1,1,1));

      color.x = imageAccumd[index].x*255.0;
      color.y = imageAccumd[index].y*255.0;
      color.z = imageAccumd[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }

	  if(color.x < 0)
		  color.x = 0;

	  if(color.y < 0)
		  color.y = 0;

	  if(color.z < 0)
		  color.z = 0;

      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

__global__ void constructRayPool(ray* rayPool, cameraData cam, float time)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * cam.resolution.x);

	// for anti-aliasing: use jittered ray cast instead.
	ray r = jitteredRaycastFromCameraKernel(cam.resolution, time, (float)x, (float)y, cam.position, cam.view, cam.up, cam.fov);
	r.isTerminated = false;
	r.pixelID = index;
	r.attenuation = vec3(1,1,1);
	rayPool[index] = r;
}


// Loop through geometry and test against ray.
// Returns FLT_MAX if no object is intersected with the ray, else returns t such that isectPoint = P + Dt 
// Input:  staticGeom* geoms: array of geometry in the scene
//         int numberOfGeoms: number of geoms in the scene
//		   ray r: the ray that is to be intersected with all the geometry
//		   int geomIdToSkip indicates the id of the geometry to skip intersection test. This prevents self-intersection for meshes.
// Output: vec3 isectPoint: holders the intersection point.
//		   vec3 isectNormal: holds the normal at the intersection point.
//		   int matId: the index of the material of the intersected geometry
//		   int geomId: the index of the geom that was hit
__device__ float intersectionTest(staticGeom* geoms, int numberOfGeoms, ray r, int geomIdToSkip, vec3 &isectPoint, vec3 &isectNormal, int &matId, int& geomId)
{
	float t = FLT_MAX;
	geomId = -1;

	// testing intersections
	for (int i = 0 ; i < numberOfGeoms ; ++i)
	{
		if (i == geomIdToSkip)
			continue;

		if (geoms[i].type == GEOMTYPE::SPHERE)
		{
			// do sphere intersection
			vec3 isectPointTemp = vec3(0,0,0);
			vec3 isectNormalTemp = vec3(0,0,0);

			float dist = sphereIntersectionTest(geoms[i], r, isectPointTemp, isectNormalTemp);

			if (dist < t && dist != -1)
			{
				t = dist;
				isectPoint = isectPointTemp;
				isectNormal = isectNormalTemp;
				matId = geoms[i].materialid;
				geomId = i;
			}
		}
		else if (geoms[i].type == GEOMTYPE::CUBE)
		{
			// do cube intersection
			vec3 isectPointTemp = vec3(0,0,0);
			vec3 isectNormalTemp = vec3(0,0,0);

			float dist = boxIntersectionTest(geoms[i], r, isectPointTemp, isectNormalTemp);

			if (dist < t && dist != -1)
			{
				t = dist;
				isectPoint = isectPointTemp;
				isectNormal = isectNormalTemp;
				matId = geoms[i].materialid;
				geomId = i;
			}
		}
		else if (geoms[i].type == GEOMTYPE::MESH)
		{
			// iterate through each triangle and find the miniumum t
			//triangleIntersectionTest(const glm::vec3& v1, const glm::vec3& v2, const glm::vec3& v3, 
			//									   const glm::vec3& n1, const glm::vec3& n2, const glm::vec3& n3,
			//									   staticGeom geom, ray r, glm::vec3& intersectionPoint, glm::vec3& normal)

			int numIndices = geoms[i].triMesh.indicesCount;

			for (int j = 0 ; j < numIndices ; ++j)
			{
				// do triangle intersection
				unsigned int index1 = geoms[i].triMesh.indices[j]; j++;
				unsigned int index2 = geoms[i].triMesh.indices[j]; j++;
				unsigned int index3 = geoms[i].triMesh.indices[j];
				
				vec3 v1 = geoms[i].triMesh.vertices[index1];
				vec3 v2 = geoms[i].triMesh.vertices[index2];
				vec3 v3 = geoms[i].triMesh.vertices[index3];

				vec3 isectPointTemp = vec3(0,0,0);
				vec3 isectNormalTemp = vec3(0,0,0);
				
				float dist = triangleIntersectionTest(v1, v2, v3, geoms[i], r, isectPointTemp, isectNormalTemp);

				if (dist < t && dist != -1)
				{
					t = dist;
					isectPoint = isectPointTemp;
					isectNormal = isectNormalTemp;
					matId = geoms[i].materialid;
					geomId = i;
				}
			}
		}
	} 

	return t;
}

// send out shadow feeler rays and compute the tint color
// this will generate hard shadows if num shadows is set to 1
__device__ vec3 shadowFeeler(staticGeom* geoms, int numberOfGeoms, material* materials, vec3 isectPoint, vec3 isectNormal, int hitGeomId, staticGeom lightSource, float iter, int index)
{
	vec3 tint = vec3(1,1,1);
	vec3 shadowRayIsectPoint = vec3(0,0,0);
	vec3 shadowRayIsectNormal = vec3(0,0,0);
	int shadowRayIsectMatId = -1;
	float t = FLT_MAX;
	float eps = 1e-5;
	int numShadowRays = SHADOWRAY_NUM;  
	
	// number of times the shadowRays hit the light
	float hitLight = 0;	
	float maxT = 0;
	
	for (int i = 0 ; i < numShadowRays ; ++i)
	{
		vec3 lightPosition = lightSource.translation;

		if (lightSource.type == GEOMTYPE::SPHERE && numShadowRays != 1)
		{
			lightPosition = getRandomPointOnSphere(lightSource, index * iter);
		}
		else if (lightSource.type == GEOMTYPE::CUBE && numShadowRays != 1)
		{
			lightPosition = getRandomPointOnCube(lightSource, index * iter);
		}
		
		vec3 lightToIsect = lightPosition - isectPoint;
		maxT = max(maxT, length(lightToIsect));
		ray shadowRay;
		shadowRay.direction = normalize(lightToIsect);
		shadowRay.origin = isectPoint + shadowRay.direction * eps; // consider moving this in the shadow ray direction

		int geomId = -1;
		t = intersectionTest(geoms, numberOfGeoms, shadowRay, hitGeomId, shadowRayIsectPoint, shadowRayIsectNormal, shadowRayIsectMatId, geomId);

		if (t != FLT_MAX)
			hitLight += materials[shadowRayIsectMatId].emittance / (materials[shadowRayIsectMatId].emittance + eps);
	}

	tint = tint * (hitLight / (float)numShadowRays);
	return tint;
}

//Core pathtracer kernel
__global__ void pathtraceRay(ray* rayPool, glm::vec3* colors, cameraData cam, staticGeom* geoms, int numberOfGeoms, material* cudamat, 
							 int numberOfMat, int* cudalightIndex, int numberOfLights, float iter, int bounce, int blockDim1Size)
{
	// Note: For ray parallelization, these ids will not necessarily correspond to the index of the pixel that can be used for the color array.
	// So instead of using these IDs directly, store pixel index that the ray is responsible for during rayPool construction
	//int rayIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
	//int rayIdy = (blockIdx.y * blockDim.y) + threadIdx.y;
	//int rayIndex = rayIdx + (rayIdy * blockDim1Size);


	int rayIndex = (blockIdx.x * blockDim.x) + threadIdx.x;

	int rayPixelIndex = rayPool[rayIndex].pixelID;

	ray r = rayPool[rayIndex];

	vec3 isectPoint = vec3(0,0,0);
	vec3 isectNormal = vec3(0,0,0);
	int matId = -1;
	int geomId = -1;
	float t = FLT_MAX;

	if (r.isTerminated)
		return;


	// debug
	//if (r.isTerminated)
	//	colors[rayPixelIndex] = vec3(1,0,0);
	//else
	//	colors[rayPixelIndex] = vec3(0,0,0);


	if (bounce == 0)
	{
		colors[rayPixelIndex] = vec3(1,1,1);
		return;
	}

	// passing in -1 for geomToSkipId because we want to check against all geometry
	t = intersectionTest(geoms, numberOfGeoms, r, -1, isectPoint, isectNormal, matId, geomId); 

	if (t != FLT_MAX)
	{
		material isectMat = cudamat[matId];
		vec3 matColor = isectMat.color;
		vec3 rayAttenuation = r.attenuation;
		float emittance = isectMat.emittance;
		float reflectance = isectMat.hasReflective;
		
		// hit light source
		if (emittance != 0)
		{
			r.isTerminated = true;
			colors[rayPixelIndex] *= 1.5f * emittance * matColor;
		}
		else
		{
			vec3 shading = vec3(0,0,0);
	
			// compute shading and the next ray r.
			calculateBSDF(r, isectPoint, isectNormal, shading, isectMat, iter * (rayPixelIndex + rayIndex * bounce));
	
			colors[rayPixelIndex] *= rayAttenuation * shading;

			// attenuate ray
			if (reflectance == 0) // no reflectance
			{
				rayAttenuation = rayAttenuation * matColor;
			}
			else if (reflectance < 1) // partial reflectance
			{
				rayAttenuation = rayAttenuation * isectMat.specularColor;
				colors[rayPixelIndex] = (1-reflectance) * matColor;
			}
			else if (reflectance == 1)// perfect reflectance
			{
				rayAttenuation = rayAttenuation * isectMat.specularColor;
			}

			r.attenuation = rayAttenuation;

			if ((rayAttenuation.x < 0.001 && rayAttenuation.y < 0.001 && rayAttenuation.z < 0.001) || bounce == MAX_BOUNCE)
			{
				r.isTerminated = true;
				colors[rayPixelIndex] = vec3(0,0,0);
			}
		}
	}
	else
	{
		r.isTerminated = true;
		colors[rayPixelIndex] = vec3(0,0,0);
	}

	rayPool[rayIndex] = r;

	////////////////
	// Debug Code //
	////////////////

	// checking intersection
	//if (t != FLT_MAX)
	//{
	//	material isectMat = cudamat[matId];		
	//	colors[rayPixelIndex] = isectMat.color;
	//}
	
	// check if rayIdx and rayPixelIndex are correct
	//if (rayIdy > 400)
	//{
	//	colors[rayPixelIndex] = vec3(1,0,0);
	//}

	// check if stuff are being passed correctly
	//for (int i = 0 ; i < numberOfGeoms ; ++i)
	//{
	//	if (geoms[i].type == MESH)
	//	{
	//		if (geoms[i].triMesh.indicesCount == 36)
	//			colors[index] = vec3(1,0,0);

	//		if (geoms[i].triMesh.normals[23] == vec3(0, -1, 0))
	//			colors[index] = colors[index] + vec3(0,0,1);

	//		if (geoms[i].triMesh.vertices[23] == vec3(-0.5, -0.5, 0.5))
	//			colors[index] = colors[index] + vec3(0,1,0);
	//	}
	//}
}

//Core raytracer kernel
__device__ void raytraceRay(ray r, float ssratio, int index, int rayDepth, glm::vec3* colors, cameraData cam,
                            staticGeom* geoms, int numberOfGeoms, material* cudamat, int numberOfMat, int* cudalightIndex, int numberOfLights, float iter)
{
	vec3 color = vec3(0,0,0);
	vec3 reflectedColor = vec3(0,0,0);
	vec3 bgc = vec3(0,0,0);
	colors[index] = bgc;
	vec3 ambientColor = vec3(0.1, 0.1, 0.1);
  
	if (rayDepth > MAX_DEPTH) 
	{
		//bgc
		color = vec3(0,0,0); 
		return;
	}

	vec3 isectPoint = vec3(0,0,0);
	vec3 isectNormal = vec3(0,0,0);
	int matId = -1;
	int geomId = -1;
	float t = FLT_MAX;

	// passing in -1 for geomToSkipId because we want to check against all geometry
	t = intersectionTest(geoms, numberOfGeoms, r, -1, isectPoint, isectNormal, matId, geomId); 

	if (t != FLT_MAX)
	{
		material isectMat = cudamat[matId];
		
		// reflection
		if (isectMat.hasReflective > 0)
		{
			vec3 reflectedDirection = calculateReflectionDirection(isectNormal, r.direction);
			ray reflectedRay;
			reflectedRay.direction = normalize(reflectedDirection);
			reflectedRay.origin = isectPoint + isectNormal * (float)1e-5;

			// Temp: Shoot the reflected ray and see which object it hits. Use the object's color instead.
			vec3 reflectedIsectPoint = vec3(0,0,0);
			vec3 reflectedIsectNormal = vec3(0,0,0);
			int reflectedMatId = -1;
			int reflectedGeomId = -1;

			// passing in -1 for geomToSkipId because we want to check against all geometry
			float rt = intersectionTest(geoms, numberOfGeoms, reflectedRay, -1, reflectedIsectPoint, reflectedIsectNormal, reflectedMatId, reflectedGeomId);

			// recurse
			raytraceRay(reflectedRay, ssratio, index, rayDepth+1, colors, cam, geoms, numberOfGeoms, cudamat, numberOfMat, cudalightIndex, numberOfLights, iter);
			reflectedColor = colors[index];
		}


		// hit light source, so use the light source's color directly
		if (isectMat.emittance != 0)
		{
			color = color + ssratio * isectMat.color;
		}
		else
		{
			float reflectance = cudamat[matId].hasReflective;
			color = ssratio * ambientColor + reflectance * reflectedColor;

			// go through each light source and compute shading
			for (int i = 0 ; i < numberOfLights ; ++i)
			{
				staticGeom lightSource = geoms[cudalightIndex[i]];
				vec3 tint = shadowFeeler(geoms, numberOfGeoms, cudamat, isectPoint, isectNormal, geomId, lightSource, iter, index);
				//vec3 tint = shadowFeeler(geoms, numberOfGeoms, cudamat, isectPoint, isectNormal, -1, lightSource, iter, index);

				vec3 IsectToLight = normalize(lightSource.translation - isectPoint);
				vec3 IsectToEye = normalize(cam.position - isectPoint);
				vec3 lightColor = cudamat[lightSource.materialid].color;
				vec3 materialColor = cudamat[matId].color;
				float lightIntensity = cudamat[lightSource.materialid].emittance;
				float diffuseTerm = clamp(dot(isectNormal, IsectToLight), 0.0f, 1.0f);

				// calculate specular highlight
				vec3 LightToIsect = -IsectToLight;
				vec3 specReflectedRay = calculateReflectionDirection(isectNormal, LightToIsect);
				float specularTerm = pow(max(0.0f, dot(specReflectedRay, IsectToEye)), isectMat.specularExponent);
				float ks = 0.2;
				float kd = 0.7;
				float lightDist = length(IsectToLight);
				float distAttenuation = 1.0f / (lightDist * lightDist);
				
				color = color + (1 - reflectance) * tint * ssratio * (lightIntensity * lightColor * distAttenuation * 
					(kd * materialColor * diffuseTerm + ks * isectMat.specularColor * specularTerm * isectMat.specularExponent));
			}
		}
	}

	colors[index] = color;

}


// Calls Core raytracer kernel starting from eye
__global__ void launchRaytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors,
                            staticGeom* geoms, int numberOfGeoms, material* cudamat, int numberOfMat, int* cudalightIndex, int numberOfLights)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);
	
	// supersampling for anti aliasing
	vec3 color = vec3(0,0,0);
	float ss = 1.0f;

	if (ANTIALIASING_SWITCH)
		ss = 3.0f;
	
	float ssratio = 1.0f / (ss * ss);

	for(float i = 1 ; i <= ss ; i++)
	{
		for(float j = 1 ; j <= ss ; j++)
		{
			float ssx = i / ss - 1 / (ss*2.0f);
			float ssy = j / ss - 1 / (ss*2.0f);

			ray r = raycastFromCameraKernel(resolution, 0, ssx + x, ssy + y, cam.position, cam.view, cam.up, cam.fov);
			raytraceRay(r, ssratio, index, rayDepth, colors, cam, geoms, numberOfGeoms, cudamat, numberOfMat, cudalightIndex, numberOfLights, time);

			color = color + colors[index];
		}
	}

	colors[index] = color;
}

void cleanupTriMesh(thrust::device_ptr<staticGeom> geoms, int numberOfGeoms)
{
	for (int i = 0 ; i < numberOfGeoms ; ++i)
	{
		// TODO: Consider using geoms.get() first then iterate through the raw pointers.
		staticGeom sg = geoms[i];
		if (sg.type == MESH)
		{
			hipFree(sg.triMesh.indices);
			hipFree(sg.triMesh.vertices);
		}
	}
}

int getNumRays(ray* allrays, int total)
{
	int totalrays = 0;
	for (int i = 0 ; i < total ; ++i)
	{
		if (!allrays[i].isTerminated)
			totalrays++;

	}

	return totalrays;
}

// move the object a little before the rendering process.
void translateObject(geom* geoms, int geomId, int frame, int iterations, int idleFrameNum, int stopFrame, vec3 translation)
{
	if (iterations % idleFrameNum == 0 && iterations < stopFrame)
	{
		geoms[geomId].translations[frame] += translation;
		glm::mat4 transform = utilityCore::buildTransformationMatrix(geoms[geomId].translations[frame], geoms[geomId].rotations[frame], geoms[geomId].scales[frame]);
		geoms[geomId].transforms[frame] = utilityCore::glmMat4ToCudaMat4(transform);
		geoms[geomId].inverseTransforms[frame] = utilityCore::glmMat4ToCudaMat4(glm::inverse(transform));
	}
}

// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms)
{
	// increase stack size so recursion can be used.
	hipDeviceSetLimit(hipLimitStackSize, 50000*sizeof(float)); 

	int traceDepth = 1; //determines how many bounces the raytracer traces

	// set up geometry for motion blur
	if (MOTION_BLUR_SWITCH)
	{
		translateObject(geoms, 6, frame, iterations, 20, 2000, vec3(0, -0.01, 0));
	}

	// set up crucial magic
	int tileSize = 8;
	dim3 threadsPerBlock(tileSize, tileSize);
	dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
	//send image to GPU
	glm::vec3* cudaimage = NULL;
	hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
	hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);

	//package geometry and materials and sent to GPU
	int numberOfLights = 0;
	std::vector<int> lightIndices;

	staticGeom* geomList = new staticGeom[numberOfGeoms];
	for(int i=0; i<numberOfGeoms; i++)
	{
		staticGeom newStaticGeom;
		newStaticGeom.type = geoms[i].type;
		newStaticGeom.materialid = geoms[i].materialid;
		newStaticGeom.translation = geoms[i].translations[frame];
		newStaticGeom.rotation = geoms[i].rotations[frame];
		newStaticGeom.scale = geoms[i].scales[frame];
		newStaticGeom.transform = geoms[i].transforms[frame];
		newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
   
		if (newStaticGeom.type == MESH)
		{
			// need to set vertices, normals, indices, and indicesCount for newStaticGeom.triMesh
			int numVerts = geoms[i].triMesh.vertices.size();
			hipMalloc((void**)&(newStaticGeom.triMesh.vertices), numVerts*sizeof(glm::vec3));
			hipMemcpy(newStaticGeom.triMesh.vertices, &(geoms[i].triMesh.vertices[0]), numVerts*sizeof(glm::vec3), hipMemcpyHostToDevice);
		
			int numIndices = geoms[i].triMesh.indices.size();
			hipMalloc((void**)&(newStaticGeom.triMesh.indices), numIndices*sizeof(unsigned int));
			hipMemcpy(newStaticGeom.triMesh.indices, &(geoms[i].triMesh.indices[0]), numIndices*sizeof(unsigned int), hipMemcpyHostToDevice);
		
			newStaticGeom.triMesh.indicesCount = geoms[i].triMesh.indicesCount;
		}
	
		geomList[i] = newStaticGeom;
	
		if (materials[newStaticGeom.materialid].emittance != 0)
		{
			numberOfLights++;
			lightIndices.push_back(i);
		}

	}
  
	staticGeom* cudageoms = NULL;
	hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
	hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);

	// check number of materials
	int numberOfMat = 0;

	for (int i = 0 ; i < numberOfMaterials ; ++i)
	{
		numberOfMat++;
	}

	// set up lights indices to pass to cuda
	int* cudalightIndex = NULL;
	hipMalloc((void**) &cudalightIndex, numberOfLights * sizeof(int));
	hipMemcpy(cudalightIndex, &(lightIndices[0]), numberOfLights * sizeof(int), hipMemcpyHostToDevice);

	// Regaring (void**) ...
	// All CUDA API functions return an error code (or hipSuccess if no error occured). 
	// All other parameters are passed by reference. However, in plain C you cannot have references, 
	// that's why you have to pass an address of the variable that you want the return information to be stored. 
	// Since you are returning a pointer, you need to pass a double-pointer.
	material* cudamat = NULL;
	hipMalloc((void**)&cudamat, numberOfMat * sizeof(material));
	hipMemcpy(cudamat, materials, numberOfMat * sizeof(material), hipMemcpyHostToDevice);
  
	//package camera
	//Q: Why don't we need to use hipMalloc and hipMemcpy here?
	//A: During kernel execution, the value cam will be put onto GPU memory stack since this value is not being passed by reference.
	cameraData cam;
	cam.resolution = renderCam->resolution;
	cam.position = renderCam->positions[frame];
	cam.view = renderCam->views[frame];
	cam.up = renderCam->ups[frame];
	cam.fov = renderCam->fov;
	
	// LOOK: Currently assuming number of rays is the number of pixels on screen.
	int numRays = cam.resolution.x * cam.resolution.y;
	int initPoolSize = numRays;
	ray* cudaRayPool = NULL;
	
	//kernel launch
	if (PATHTRACING_SWITCH)
	{
		// construct ray pool
		hipMalloc((void**)&cudaRayPool, numRays * sizeof(ray));
		constructRayPool<<<fullBlocksPerGrid, threadsPerBlock>>>(cudaRayPool, cam, (float)iterations);

		// Trace all bounces of the ray in a BFS manner
		for(int bounce = 0; bounce <= MAX_BOUNCE; ++bounce)
		{
			// Update blockSize based on the number of rays.
			//float sqrtNumRays = ceil(sqrt((float)numRays));
			//int blockSize = (int)ceil(sqrtNumRays/(float)tileSize);
			//dim3 rayBlockPerGrid(blockSize, blockSize);
			//pathtraceRay<<<rayBlockPerGrid,threadsPerBlock>>>(cudaRayPool, cudaimage, cam, cudageoms, numberOfGeoms, cudamat, numberOfMat, cudalightIndex, numberOfLights, (float)iterations, bounce, blockSize * tileSize);
			
			// try 1D blocks...
			int totalThreadsPerBlock = tileSize * tileSize;
			int totalRayBlocksPerGrid = (int)ceil((float)numRays / (float)totalThreadsPerBlock);
			pathtraceRay<<<totalRayBlocksPerGrid,totalThreadsPerBlock>>>(cudaRayPool, cudaimage, cam, cudageoms, numberOfGeoms, cudamat, numberOfMat, cudalightIndex, numberOfLights, (float)iterations, bounce, 1);

			// Stream compaction using thrust
			if (USE_STREAM_COMPACTION)
			{
				thrust::device_ptr<ray> cudaRayPoolDevicePtr(cudaRayPool);
				thrust::device_ptr<ray> compactCudaRayPoolDevicePtr = thrust::remove_if(cudaRayPoolDevicePtr, cudaRayPoolDevicePtr + numRays, is_terminated());
			
				// pointer arithmetic to figure out the number of rays.
				numRays = compactCudaRayPoolDevicePtr.get() - cudaRayPoolDevicePtr.get();
			}

			// debugging. The size computed by getNumRays doesnt seem to match numRays above.
			//ray* allRays = new ray[(int)cam.resolution.x * (int)cam.resolution.y];
			//hipMemcpy( allRays, cudaRayPool, (int)cam.resolution.x * (int)cam.resolution.y*sizeof(ray), hipMemcpyDeviceToHost);
			//int numRays2 = getNumRays(allRays, (int)cam.resolution.x * (int)cam.resolution.y);
			//delete[] allRays;

			if (numRays < 0)
			{
				printf("Number of rays = 0. Terminating...\n");
				break;
			}
		}
	}
	else
	{
		launchRaytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms, cudamat, numberOfMat, cudalightIndex, numberOfLights);
	}
 
	// setting up previous image accumulation
	vec3* imageAccum = NULL;
	hipMalloc((void**)&imageAccum,(int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
	hipMemcpy(imageAccum, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
	//kernel launch
	sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, (float)iterations, renderCam->resolution, cudaimage, imageAccum);

	//retrieve image from GPU
	hipMemcpy( renderCam->image, imageAccum, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

	//free up stuff, or else we'll leak memory like a madman
	//thrust::device_ptr<staticGeom> cudageomsPtr(cudageoms); // TODO: Figure out a better way to free cudageoms->triMesh
	//cleanupTriMesh(cudageomsPtr, numberOfGeoms);
	hipFree( cudaimage );
	hipFree( cudageoms );
	hipFree( imageAccum );
	hipFree( cudamat );
	hipFree( cudalightIndex );

	if (PATHTRACING_SWITCH)
	{
		hipFree( cudaRayPool );
	}

	delete[] geomList;

	// make certain the kernel has completed
	hipDeviceSynchronize();

	checkCUDAError("Kernel failed!");
}

