#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>
#include "glm/glm.hpp"

void checkCUDAError(const char *msg) {
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
		exit(EXIT_FAILURE); 
	}
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
	int index = x + (y * resolution.x);

	thrust::default_random_engine rng(hash(index*time));
	thrust::uniform_real_distribution<float> u01(0,1);

	return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//Function that does the initial raycast from the camera given a float defined pixel. Allows pixels to be defined with subpixel resolution easily.
//20% faster than provided code.
__host__ __device__ ray raycastFromCamera(glm::vec2 resolution, float x, float y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){

	ray r;
	r.origin = eye;
	glm::vec3 right = glm::cross(view, up);

	//float d = 1.0f; use a viewing plane of 1 distance 
	glm::vec3 pixel_location = /* d* */(view + (2*x/resolution.x-1)*right*glm::tan(glm::radians(fov.x)) 
		- (2*y/resolution.y-1)*up*glm::tan(glm::radians(fov.y)));

	r.direction = glm::normalize(pixel_location);

	return r;

}


//Scales the entire image by a float scale factor. Makes averaging trivial
__global__ void scaleImageIntensity(glm::vec2 resolution, glm::vec3* image, float sf)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	if(x<resolution.x && y<resolution.y){
		image[index] = sf*image[index];
	}
}

//Takes each ray's pixel assignment and casts a randomized ray through the pixel
__global__ void raycastFromCameraKernel(int seed, int frame, cameraData cam, renderOptions rconfig, rayState* cudaraypool, int rayPoolSize)
{	

	int blockId   = blockIdx.y * gridDim.x + blockIdx.x;			 	
	int rIndex = blockId * blockDim.x + threadIdx.x; 
	if(rIndex < rayPoolSize){
		//read from global mem
		rayState rstate = cudaraypool[rIndex];
		int pixelIndex = rstate.index;
		if(pixelIndex >= 0){
			int x = pixelIndex % int(cam.resolution.x);
			int y = (pixelIndex - x)/int(cam.resolution.x);

			//Reset other fields
			rstate.T = glm::vec3(1,1,1);
			rstate.matIndex = -1;
			if(rconfig.antialiasing){
		thrust::default_random_engine rng(hash(seed*rIndex));//TODO: Improve randomness
		thrust::uniform_real_distribution<float> u01(-0.5,0.5);
				rstate.r =raycastFromCamera(cam.resolution, x+u01(rng), y+u01(rng), cam.position, cam.view, cam.up, cam.fov);
			}else{
				rstate.r =raycastFromCamera(cam.resolution, x, y, cam.position, cam.view, cam.up, cam.fov);
			}
			rstate.bounceType = PRIMARY;
			//write back to global mem
			cudaraypool[rIndex] = rstate;

		}
	}

}

//Takes the number of rays requested by each pixel from the pool and allocates them stocastically from a single random number
//xi1 is a uniformly distributed random number from 0 to 1
__global__ void allocateRayPool(float xi1, renderOptions rconfig, cameraData cam, glm::vec3* cudaimage, rayState* cudaraypool, int numRays)
{
	//1D blocks and 2D grid

	int blockId   = blockIdx.y * gridDim.x + blockIdx.x;			 	
	int rIndex = blockId * blockDim.x + threadIdx.x; 

	if(rIndex < numRays){//Thread index range check

		int numPixels = cam.resolution.x*cam.resolution.y;

		//Allocate all rays stochastically
		if(rconfig.stocasticRayAssignment){
			float P = float(numPixels)/numRays;//compute stochastic interval
			int start =  floor(xi1*numPixels);
			cudaraypool[rIndex].index = ((int)(start + P*rIndex) % numPixels);
		}else{
			if(rIndex < numPixels)
				cudaraypool[rIndex].index = rIndex;
			else
				cudaraypool[rIndex].index = -1;
		}
	}
}

__global__ void displayRayCounts(cameraData cam, renderOptions rconfig, glm::vec3* cudaimage, rayState* cudaraypool, int numRays, float maxScale)
{
	int blockId   = blockIdx.y * gridDim.x + blockIdx.x;			 	
	int rIndex = blockId * blockDim.x + threadIdx.x; 

	if(rIndex < numRays){//Thread index range check
		int pixelIndex = cudaraypool[rIndex].index;
		if(pixelIndex >= 0)
		{
			float scale = clamp(1.0f/maxScale, 0.0f, 1.0f);
			cudaimage[pixelIndex] += scale*glm::vec3(1,1,1);
		}
	}

}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);
	if(x<=resolution.x && y<=resolution.y){
		image[index] = glm::vec3(0,0,0);
	}
}


//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image, float scaleFactor){

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	if(x<=resolution.x && y<=resolution.y){

		glm::vec3 color;      
		color.x = image[index].x*255.0*scaleFactor;
		color.y = image[index].y*255.0*scaleFactor;
		color.z = image[index].z*255.0*scaleFactor;

		//Clamp
		if(color.x>255){
			color.x = 255;
		}

		if(color.y>255){
			color.y = 255;
		}

		if(color.z>255){
			color.z = 255;
		}

		// Each thread writes one pixel location in the texture (textel)
		PBOpos[index].w = 0;
		PBOpos[index].x = color.x;     
		PBOpos[index].y = color.y;
		PBOpos[index].z = color.z;
	}
}

__global__ void traceRayFirstHit(cameraData cam, renderOptions rconfig, float time, int bounce, glm::vec3* colors, 
								 rayState* raypool, int numRays, staticGeom* geoms, int numberOfGeoms, material* materials, int numberOfMaterials)
{
	//Compute ray index
	int blockId   = blockIdx.y * gridDim.x + blockIdx.x;			 	
	int rIndex = blockId * blockDim.x + threadIdx.x; 


	//Pixel index of -1 indicates the ray's contribution has been recorded and is no longer in flight
	if(rIndex < numRays)
	{
		//Thread has a ray, check if ray has a pixel
		int pixelIndex = raypool[rIndex].index;
		if(pixelIndex >= 0 && pixelIndex < (int)cam.resolution.x*(int)cam.resolution.y)
		{
			//valid pixel index
			ray r = raypool[rIndex].r;

			float dist;
			glm::vec3 intersectionPoint;
			glm::vec3 normal;
			int ind = firstIntersect(geoms, numberOfGeoms, r, intersectionPoint, normal, dist);
			if(rconfig.mode == FIRST_HIT_DEBUG){
				if(ind >= 0)
					colors[pixelIndex] += materials[geoms[ind].materialid].color;
				else
					colors[pixelIndex] += rconfig.backgroundColor;
			}else if(rconfig.mode == NORMAL_DEBUG){
				colors[pixelIndex] += glm::abs(normal);
			}
		}	
	}
}


//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void traceRay(cameraData cam, renderOptions rconfig, float time, int bounce, glm::vec3* colors, 
						 rayState* raypool, int numRays, staticGeom* geoms, int numberOfGeoms, material* materials, int numberOfMaterials)
{
	//Compute ray index
	int blockId   = blockIdx.y * gridDim.x + blockIdx.x;			 	
	int rIndex = blockId * blockDim.x + threadIdx.x; 


	//Pixel index of -1 indicates the ray's contribution has been recorded and is no longer in flight
	if(rIndex < numRays)
	{
		//Thread has a ray, check if ray has a pixel
		int pixelIndex = raypool[rIndex].index;
		if(pixelIndex >= 0 && pixelIndex < (int)cam.resolution.x*(int)cam.resolution.y)
		{
			//valid pixel index
			rayState rstate = raypool[rIndex];
			//Check if ray is still useful
			if(rstate.T.x > rconfig.minT || rstate.T.y > rconfig.minT || rstate.T.z > rconfig.minT)
			{
				//ray still has transmitance worth considering

				//Find first collision
				float dist;
				glm::vec3 intersectionPoint;
				glm::vec3 normal;
				int ind = firstIntersect(geoms, numberOfGeoms, rstate.r, intersectionPoint, normal, dist);

				if(ind >= 0){
					//we hit something!

					//calculate transmission through material
					glm::vec3 absorbtionCoeff;
					if(rstate.matIndex >= 0 )
						absorbtionCoeff = materials[rstate.matIndex].absorptionCoefficient;
					else
						absorbtionCoeff = rconfig.airAbsorbtion;

					rstate.T *= calculateTransmission(absorbtionCoeff, dist);

					//Transmission computed, now let's check on what we hit. This is where code will diverge quite a bit

					//Check if it's a light
					material m = materials[geoms[ind].materialid];
					if(m.emittance > 0)
					{
						//hit a light source. Light it up.
						if(rconfig.mode == TRACEDEPTH_DEBUG){
							colors[pixelIndex] += bounce/float(rconfig.traceDepth)*glm::vec3(1,1,1);
							rstate.index = -1;//retire ray
						}else if(rconfig.mode == PATHTRACE){
							colors[pixelIndex] += rstate.T*m.emittance*m.color;
							rstate.index = -1;//retire ray
						}
					}else{
						if(bounce < rconfig.traceDepth - 1){
							//if we have more bounces to do, Bounce ray. 

							//TODO: Improve randomness with point sets?
							thrust::default_random_engine rng(hash(time*rIndex));
							thrust::uniform_real_distribution<float> u01(0,1);
							bounceRay(rstate, rconfig, intersectionPoint, normal, materials, geoms[ind].materialid, u01(rng), u01(rng), u01(rng));

						}else{
							//This was the last bounce. 
							if(rconfig.mode == TRACEDEPTH_DEBUG)
							{
								colors[pixelIndex] += bounce/float(rconfig.traceDepth)*glm::vec3(1,1,1);
								rstate.index = -1;//retire ray
							}else if(rconfig.mode == PATHTRACE)
							{
								//This photon didn't come from anything
								rstate.index = -1;

							}
						}
					}
				}else{
					//How could you miss it was right in front of you!!
					//retire ray, add global illumination compnent

					if(rconfig.mode == TRACEDEPTH_DEBUG){
						colors[pixelIndex] += bounce/float(rconfig.traceDepth)*glm::vec3(1,1,1);
						rstate.index = -1;//retire ray
					}else if(rconfig.mode == PATHTRACE){
						//Compute global illumination component, we've hit the sky
						if(rstate.bounceType == DIFFUSE || rstate.bounceType == TRANSMIT){
						float globalLightDot = clamp(-glm::dot(rstate.r.direction, rconfig.globalLightDirection),0.0f,1.0f);
							colors[pixelIndex] += rstate.T*rconfig.globalLightColor*rconfig.globalLightIntensity*globalLightDot;
						}else{
							//Primary or specular reflection, display color
							colors[pixelIndex] += rstate.T*rconfig.backgroundColor;
						}
						rstate.index = -1;//retire ray
					}
				}
			}else{
				//Ray no longer transmits any useful info
				//Retire it
				rstate.index = -1;
			}
			//Write back
			raypool[rIndex] = rstate;
		}	
	}
}


//TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam,  renderOptions* rconfig, int frame, int iterations, int frameFilterCounter, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){

	int traceDepth = rconfig->traceDepth; //determines how many bounces the raytracer traces
	int numPixels = renderCam->resolution.x*renderCam->resolution.y;
	int rayPoolSize = (int) ceil(float(numPixels)*rconfig->rayPoolSize);

	// set up crucial magic
	int tileSize = 8;
	dim3 threadsPerBlockByPixel(tileSize, tileSize);
	dim3 fullBlocksPerGridByPixel((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));

	// Set up a 2D grid
	// Fill up rows before adding more
	//TODO: Improve resource allocation. Slipping over once will create a lot of wasted blocks
	int blockSize = 64;
	dim3 threadsPerBlockByRay(blockSize);
	int blockCount = (int)ceil(float(rayPoolSize)/float(blockSize));

	dim3 fullBlocksPerGridByRay;
	int maxGridX = 65535;//TODO: get this dynamically
	if(blockCount > maxGridX){
		fullBlocksPerGridByRay = dim3(maxGridX, (int)ceil( blockCount / float(maxGridX)));
	}else{
		fullBlocksPerGridByRay = dim3(blockCount);
	}

	//send image to GPU
	glm::vec3* cudaimage = NULL;
	hipMalloc((void**)&cudaimage, numPixels*sizeof(glm::vec3));
	hipMemcpy( cudaimage, renderCam->image, numPixels*sizeof(glm::vec3), hipMemcpyHostToDevice);

	//package geometry and materials and sent to GPU
	staticGeom* geomList = new staticGeom[numberOfGeoms];
	for(int i=0; i<numberOfGeoms; i++){
		staticGeom newStaticGeom;
		newStaticGeom.type = geoms[i].type;
		newStaticGeom.materialid = geoms[i].materialid;
		newStaticGeom.translation = geoms[i].translations[frame];
		newStaticGeom.rotation = geoms[i].rotations[frame];
		newStaticGeom.scale = geoms[i].scales[frame];
		newStaticGeom.transform = geoms[i].transforms[frame];
		newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
		geomList[i] = newStaticGeom;
	}

	///Allocations
	staticGeom* cudageoms = NULL;
	hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
	hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);

	material* cudamaterials = NULL;
	hipMalloc((void**)&cudamaterials, numberOfMaterials*sizeof(material));
	hipMemcpy( cudamaterials, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

	rayState* cudaraypool = NULL;
	hipMalloc((void**)&cudaraypool, rayPoolSize*sizeof(rayState));


	//package camera
	cameraData cam;
	cam.resolution = renderCam->resolution;
	cam.position = renderCam->positions[frame];
	cam.view = renderCam->views[frame];
	cam.up = renderCam->ups[frame];
	cam.fov = renderCam->fov;

	///Prep image
	if(!rconfig->frameFiltering || frameFilterCounter <= 1)
	{
		clearImage<<<fullBlocksPerGridByPixel, threadsPerBlockByPixel>>>(renderCam->resolution, cudaimage);
		frameFilterCounter = 1;

	}
	//else{
	//	scaleImageIntensity<<<fullBlocksPerGridByPixel, threadsPerBlockByPixel>>>(renderCam->resolution, cudaimage, (float)(frameFilterCounter-1));
	//}


	//Figure out which rays should go to which pixels.
	thrust::default_random_engine rng(hash(iterations*frameFilterCounter+iterations));
	thrust::uniform_real_distribution<float> u01(0,1);
	allocateRayPool<<<fullBlocksPerGridByRay, threadsPerBlockByRay>>>(u01(rng), *rconfig, cam, cudaimage, cudaraypool, rayPoolSize);

	switch(rconfig->mode)
	{
	case TRACEDEPTH_DEBUG:
	case PATHTRACE:
		raycastFromCameraKernel<<<fullBlocksPerGridByRay, threadsPerBlockByRay>>>(iterations, frame, cam, *rconfig, cudaraypool, rayPoolSize);

		for(int bounce = 0; bounce < traceDepth; bounce++)
		{
			traceRay<<<fullBlocksPerGridByRay, threadsPerBlockByRay>>>(cam, *rconfig, iterations, bounce, cudaimage, 
				cudaraypool, rayPoolSize, cudageoms, numberOfGeoms, cudamaterials, numberOfMaterials);
			/*if(rconfig.streamCompaction)
			{
			int rayPoolSize = raypoolCompaction(&cudaraypool, rayPoolSize);

			blockCount = (int)ceil(float(rayPoolSize)/float(blockSize));

			dim3 fullBlocksPerGridByRay;
			if(blockCount > maxGridX){
			fullBlocksPerGridByRay = dim3(maxGridX, (int)ceil( blockCount / float(maxGridX)));
			}else{
			fullBlocksPerGridByRay = dim3(blockCount);
			}
			}*/
		}

		break;
	case RAYCOUNT_DEBUG:
		displayRayCounts<<<fullBlocksPerGridByRay, threadsPerBlockByRay>>>(cam, *rconfig, cudaimage, cudaraypool, rayPoolSize,ceil(float(rayPoolSize)/numPixels));
		break;

	case NORMAL_DEBUG:
	case FIRST_HIT_DEBUG:
		raycastFromCameraKernel<<<fullBlocksPerGridByRay, threadsPerBlockByRay>>>(iterations, frame, cam, *rconfig, cudaraypool, rayPoolSize);

		traceRayFirstHit<<<fullBlocksPerGridByRay, threadsPerBlockByRay>>>(cam, *rconfig, iterations, 0, cudaimage, 
			cudaraypool, rayPoolSize, cudageoms, numberOfGeoms, cudamaterials, numberOfMaterials);

		break;
	}


	//if(rconfig->frameFiltering)
	//{
	//	scaleImageIntensity<<<fullBlocksPerGridByPixel, threadsPerBlockByPixel>>>(renderCam->resolution, cudaimage, 1.0f/(frameFilterCounter));
	//}


	//retrieve image from GPU before drawing overlays and writing to screen
	hipMemcpy( renderCam->image, cudaimage, numPixels*sizeof(glm::vec3), hipMemcpyDeviceToHost);

	//TODO: Draw any debug overlays here



	//Draw to screen
	sendImageToPBO<<<fullBlocksPerGridByPixel, threadsPerBlockByPixel>>>(PBOpos, renderCam->resolution, cudaimage, 1.0f/float(frameFilterCounter));


	//free up stuff, or else we'll leak memory like a madman
	hipFree( cudaimage );
	hipFree( cudageoms );
	hipFree( cudamaterials );
	hipFree( cudaraypool );
	delete [] geomList;

	// make certain the kernel has completed 
	hipDeviceSynchronize();
	checkCUDAError("Kernel failed!");
}
