#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>
#include "glm/glm.hpp"

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

////Kernel that does the initial raycast from the camera.
//__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){
//   
//  int index = x + (y * resolution.x);
//   
//  thrust::default_random_engine rng(hash(index*time));
//  thrust::uniform_real_distribution<float> u01(0,1);
//  
//  //standard camera raycast stuff
//  glm::vec3 E = eye;
//  glm::vec3 C = view;
//  glm::vec3 U = up;
//  float fovx = fov.x;
//  float fovy = fov.y;
//  
//  float CD = glm::length(C);
//  
//  glm::vec3 A = glm::cross(C, U);
//  glm::vec3 B = glm::cross(A, C);
//  glm::vec3 M = E+C;
//  glm::vec3 H = (A*float(CD*tan(fovx*(PI/180))))/float(glm::length(A));
//  glm::vec3 V = (B*float(CD*tan(-fovy*(PI/180))))/float(glm::length(B));
//  
//  float sx = (x)/(resolution.x-1);
//  float sy = (y)/(resolution.y-1);
//  
//  glm::vec3 P = M + (((2*sx)-1)*H) + (((2*sy)-1)*V);
//  glm::vec3 PmE = P-E;
//  glm::vec3 R = E + (float(200)*(PmE))/float(glm::length(PmE));
//  
//  glm::vec3 direction = glm::normalize(R);
//  //major performance cliff at this point, TODO: find out why!
//  ray r;
//  r.origin = eye;
//  r.direction = direction;
//  return r;
//}

//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){
  
  ray r;
  float theta = fov.x*PI/180.0f;
  float phi = fov.y*PI/180.0f;

  glm::vec3 A = glm::cross(view,up);
  glm::vec3 B = glm::cross(A,view);
  glm::vec3 M = eye + view;
  glm::vec3 H = glm::normalize(A)*glm::length(view)*tan(theta);
  glm::vec3 V = glm::normalize(B)*glm::length(view)*tan(phi);

  float sx= -1.0f; //(float)x/(resolution.x-1);
  float sy = -1.0f;//1.0 - (float)y/(resolution.y-1);
  
  thrust::default_random_engine rng(hash(43231*time));
  thrust::uniform_real_distribution<float> u01(-0.95,0.95);

  while( sx<=0.0f || sx>=1.0f || sx<=0.0f || sy>=1.0f)
	{
		float xrand = x + u01(rng);
		float yrand = y + u01(rng);
		sx = xrand/(resolution.x-1);
		sy = 1.0f - yrand/ (resolution.y-1);
	}
  glm::vec3 P = M + (2*sx-1)*H + (2*sy - 1)*V;
  r.origin = eye;
  r.direction = glm::normalize(P-eye);
  return r;
}


//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image, ray* rays,int iterations){
  
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	int x = rays[index].pixelIndex.x;
	int y = rays[index].pixelIndex.y;
	int pixelIndex = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;    
      color.x = image[pixelIndex].x*255.0/iterations;
      color.y = image[pixelIndex].y*255.0/iterations;
      color.z = image[pixelIndex].z*255.0/iterations;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;     
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;

  }
}

__device__ bool isLight(int objId, int* lights, int numberOfLights)
{
	for (int i=0; i<numberOfLights; ++i)
		if (lights[i] == objId)
			return true;
	return false;
}

//Kernel that blacks out a given image buffer
__global__ void clearActiveRays(glm::vec2 resolution, ray* rays, glm::vec3* image){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	int x = rays[index].pixelIndex.x;
	int y = rays[index].pixelIndex.y;
	int pixelIndex = x + (y * resolution.x);
    //int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    //int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    //int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y ){
		if (!rays[index].active)
			image[pixelIndex]+= rays[index].rayColor;
    }
}

//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, float bounce, cameraData cam, int rayDepth, glm::vec3* colors, 
                            staticGeom* geoms, int numberOfGeoms, material* materials, int numberOfMaterials, int* lights, int numberOfLights,ray* rays){

  //int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  //int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  //int index = x + (y * resolution.x);
	
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  int x=-1;
  int y=-1;
  ray r;

  if (bounce==1)
  {
	y = (int) (index/(int)resolution.x);
	x = (int) (index%(int)resolution.x);

	r = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov);
	r.active = true;
	r.pixelIndex = glm::vec2(x,y);
	r.rayColor = glm::vec3(1,1,1);
	rays[index].rayColor = glm::vec3(1,1,1);
	rays[index].pixelIndex = r.pixelIndex;
	//rays[index].rayColor = glm::vec3(1,1,1); //White initially
	//rays[index].origin = r.origin;
	//rays[index].direction = r.direction;
	//rays[index].active = r.active;
	//rays[index].pixelIndex = r.pixelIndex;
  }
  else
  {
	  r = rays[index];
	  x = r.pixelIndex.x;
	  y = r.pixelIndex.y;
	  index = x + (y * resolution.x);
  }

  if((x<=resolution.x && y<=resolution.y && r.active)){
	glm::vec3 intersectionPoint;
	glm::vec3 intersectionNormal;

	int objId = findNearestGeometricIntersection(r,intersectionPoint,intersectionNormal,geoms,numberOfGeoms);

	if (objId == -1)
	{
		//r.active = false;
		rays[index].active = false;
		rays[index].rayColor = glm::vec3(0,0,0);
		return;
	}
	material mtl = materials[geoms[objId].materialid];
	/*if (isLight(objId,lights,numberOfLights))*/
	if (objId == 8)
	{
		//r.active = false;
		rays[index].active = false;
		rays[index].rayColor.x *= mtl.color.x*mtl.emittance;
		rays[index].rayColor.y *= mtl.color.y*mtl.emittance;
		rays[index].rayColor.z *= mtl.color.z*mtl.emittance;
		return;
	}

	glm::vec3 emittedColor;
	glm::vec3 unabsorbedColor;
	int bsdf = calculateBSDF(r,intersectionPoint,intersectionNormal,emittedColor,colors[index],unabsorbedColor,mtl,bounce*time*index);
	
	if (bsdf == 0)
	{		
		r.rayColor.x *= mtl.color.x;
		r.rayColor.y *= mtl.color.y;
		r.rayColor.z *= mtl.color.z;
	}
	else if(bsdf == 1)
	{
		r.rayColor.x *= mtl.specularColor.x;
		r.rayColor.y *= mtl.specularColor.y;
		r.rayColor.z *= mtl.specularColor.z;
	}
	else if (bsdf == 2)
	{
		r.rayColor.x = mtl.color.x;
		r.rayColor.y = mtl.color.y;
		r.rayColor.z = mtl.color.z;
	}

	rays[index].origin = r.origin + 0.001f*r.direction;
	rays[index].direction = r.direction;
	rays[index].active = r.active;
	rays[index].pixelIndex = r.pixelIndex;
	rays[index].rayColor = r.rayColor;
	//colors[index] = glm::abs(r.direction);
    //colors[index] = generateRandomNumberFromThread(resolution, time, x, y);
   }
	
}

//__global__ void createBinaryArray(ray* rays,int* activeRaysArray, int lastIndex)
//{
//  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
//  int y = (blockIdx.y * blockDim.y) + threadIdx.y;	
//}
//
//__global__ void streamCompact(ray* rays, int* activeRaysArray int lastIndex)
//{
//	int i;
//}

//TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){
  
  int traceDepth = 1; //determines how many bounces the raytracer traces

  // set up crucial magic
  int numberOfThreads = (int)(renderCam->resolution.x)*(int)(renderCam->resolution.y);
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize*tileSize);
  dim3 fullBlocksPerGrid ( (int) ceil( (float)numberOfThreads/(tileSize*tileSize)));
 /* dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
*/
  //send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
   //package lights
  std::vector<int> lightsVector;

  ray* rays = new ray[ (int)renderCam->resolution.x*(int)renderCam->resolution.y];
  ray* cudarays = NULL;
  hipMalloc((void**)&cudarays, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(ray));
  hipMemcpy( cudarays, rays, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(ray), hipMemcpyHostToDevice);

  //package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;

	if (materials[geoms[i].materialid].emittance > 0.0f)
		lightsVector.push_back(i);
  }
  

  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);
  
  material* cudamaterials = NULL;
  hipMalloc((void**)&cudamaterials, numberOfMaterials*sizeof(material));
  hipMemcpy( cudamaterials, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

  int numberOfLights = lightsVector.size();
  int* cudalights = NULL;
  hipMalloc( (void**)&cudalights, numberOfLights*sizeof(int));
  hipMemcpy(cudalights,&lightsVector,numberOfLights*sizeof(int),hipMemcpyHostToDevice);
  
  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;

  //kernel launches
  for(int bounce = 1; bounce <= 3; ++bounce)
  {
	raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, (float)bounce, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms, cudamaterials, numberOfMaterials, cudalights,numberOfLights,cudarays);
  }
  clearActiveRays<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution,cudarays, cudaimage);
  
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage,cudarays,iterations);

  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);
  //clearImage<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, cudaimage);
  //free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  hipFree( cudamaterials );
  hipFree(cudalights);
  hipFree(cudarays);
  delete [] geomList;
  delete [] rays;

  // make certain the kernel has completed 
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}
