#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){
	
	ray r;

	//ray creation from camear stuff to be used in raycastFromCameraKernel
	glm::vec3 M = eye + view;	//center of screen

	//project screen to world space
	glm::vec3 A = glm::cross(view, up);
	glm::vec3 B = glm::cross(A, view);

	float C = glm::length(view);

	float phi = fov.y/180.0f * PI;		//convert to radians
	B = glm::normalize(B);
	glm::vec3 V = C * tan(phi) * B;

	float theta = fov.x/180.0f * PI;
	A = glm::normalize(A);
	glm::vec3 H = C * tan(theta) * A;

	//find the world space coord of the pixel
	float sx = (float)x / (resolution.x-1.0f);
	float sy = (float)y / (resolution.y-1.0f);

	glm::vec3 P = M + H * (2.0f * sx - 1.0f) + V * (1.0f - 2.0f * sy);

	r.origin = eye;
	r.direction = glm::normalize(P - eye);

	return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image, float iterations){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  //iterations = 1;

  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;
      color.x = image[index].x*255.0 / iterations;
      color.y = image[index].y*255.0 / iterations;
      color.z = image[index].z*255.0 / iterations;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}


//Does intersection on all of the objects and returns length of closest intersection
__host__ __device__ float testGeomIntersection(staticGeom* geoms, int numberOfGeoms, ray& r, glm::vec3& intersectionPoint, glm::vec3& normal, int& objID){

	float len = FLT_MAX;	
	float tempLen = -1;
	glm::vec3 tempIntersection;
	glm::vec3 tempNormal;
	
	//check for interesction
	for(int geomInd = 0; geomInd<numberOfGeoms; ++geomInd){
			
		if(geoms[geomInd].type == CUBE){
			tempLen = boxIntersectionTest(geoms[geomInd], r, tempIntersection, tempNormal);
		}

		else if (geoms[geomInd].type == SPHERE){
			tempLen = sphereIntersectionTest(geoms[geomInd], r, tempIntersection, tempNormal);
		}
			
		else if(geoms[geomInd].type == MESH){
				
		}
							
		//if intersection occurs and object is in front of previously intersected object
		if(tempLen != -1 && tempLen < len){
			len =tempLen;
			intersectionPoint = tempIntersection;
			normal = tempNormal;
			objID = geomInd;
		}
	}

	return len;

}

//creates and stores first bounce rays, always at depth 1
__global__ void createRay(cameraData cam, staticGeom* geoms, int numberOfGeoms, material* materials, int numLights, int* lightID, rayBounce* firstPass){

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * cam.resolution.x);
	
	//do camera cast bullshit
	glm::vec3 intersection;
	glm::vec3 normal;


}


//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors,
                            staticGeom* geoms, int numberOfGeoms, material* materials, int numLights, int* lightID, rayBounce* firstPass){

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	glm::vec3 intersection;
	glm::vec3 normal;
	glm::vec3 surfColor;

	int currDepth = 1;

	if((x<=resolution.x && y<=resolution.y)){

		ray firstRay = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov); 
		glm::vec3 finalColor(1,1,1);
		
		while(currDepth <= rayDepth){

#pragma region setup
			//DOF setup thing
			float focalLength = cam.focalLength;

			float aperture = cam.aperture;
		
			glm::vec3 focalPoint = firstRay.origin + focalLength * firstRay.direction;

			//jitter camera
			glm::vec3 jitterVal = 2.0f * aperture * generateRandomNumberFromThread(resolution, time, x, y);
			jitterVal -= glm::vec3(aperture);
			firstRay.origin += jitterVal;

			//find new direction
			firstRay.direction = glm::normalize(focalPoint - firstRay.origin);

			//antialias sample per pixel
			jitterVal = generateRandomNumberFromThread(resolution, time, x, y);
			jitterVal -= glm::vec3(0.5f, 0.5f, 0.5f);
			firstRay.direction += 0.0015f* jitterVal; 

			//firstPass[index] = firstRay;
#pragma endregion setup

			//do intersection test
			int objID = -1;

			float len = testGeomIntersection(geoms, numberOfGeoms, firstRay, intersection, normal, objID);

			//if no intersection, return
			if(objID == -1){
				finalColor *= 0.0f;
				break;	
			}
		
			int matID = geoms[objID].materialid;
			surfColor = materials[matID].color;

			//check if you intersected with light, if so, just return light color
			if(materials[matID].emittance > 0){
				finalColor *= surfColor;
				break;
			}


	#pragma region lightAndShadow
			glm::vec3 diffuse(0,0,0);
			glm::vec3 phong(0,0,0);

			//do light and shadow computation
			for(int i = 0; i < numLights; ++i){

				int lightGeomID = lightID[i];
				glm::vec3 lightPos;
				glm::vec3 lightColor = materials[geoms[lightGeomID].materialid].color;

				//find a random point on the light
				if(geoms[lightGeomID].type == CUBE){
					lightPos = getRandomPointOnCube(geoms[lightGeomID], time);		//CHANGE TO TIME!
				}
				else if(geoms[lightGeomID].type == SPHERE){
					lightPos = getRandomPointOnSphere(geoms[lightGeomID], time);	//CHANGE TO TIME!
				}

				//find vector from intersection to point on light
				glm::vec3 L = lightPos - intersection;
				float distToLight = glm::length(L);
				L = glm::normalize(L);

				//check if in shadow
				objID = -1;
				ray shadowFeeler; 
				shadowFeeler.direction = L;
				shadowFeeler.origin = intersection + 0.0001f*L;		//offset origin a little bit so it doesn't self intersect
			
				glm::vec3 shadowNormal; glm::vec3 shadowIntersection;
				len = testGeomIntersection(geoms, numberOfGeoms, shadowFeeler, shadowIntersection, shadowNormal, objID);

				//if intersection occured and intersection is in between the intersection point and the light position
				if(objID != -1 && len < distToLight){
				
					if(materials[geoms[objID].materialid].emittance == 0){		//only cast shadow if we intersected with object that is not a light
						//color is ambient color
						finalColor = glm::vec3(0,0,0);
						continue;
					}
				}

				//do diffuse calculation
				diffuse += glm::clamp(glm::dot(L, normal), 0.0f, 1.0f) * surfColor * lightColor;
			
				//clamp diffuse to surface color
				diffuse.x = clamp(diffuse.x, 0.0f, surfColor.x);
				diffuse.y = clamp(diffuse.y, 0.0f, surfColor.y);
				diffuse.z = clamp(diffuse.z, 0.0f, surfColor.z);

				//specular
				if(materials[matID].specularExponent != 0){
					glm::vec3 R = glm::normalize( -L - 2.0f*glm::dot(-L, normal) *normal);
					glm::vec3 V = -firstRay.direction;			//already normalized
			
					phong += materials[matID].specularColor * 
							pow(glm::clamp(glm::dot(R, V), 0.0f, 1.0f), materials[matID].specularExponent) * lightColor;
					//phong *= 0.5f;
					//diffuse *= 0.9f;
				}

			}
	#pragma endregion lightAndShadow

			//check for reflection
			if(materials[matID].hasReflective == 1){
				//reflect
				firstRay.direction = glm::normalize(firstRay.direction - 2.0f*glm::dot(firstRay.direction, normal)*normal);
				
				//offsect a little to prevent intersection
				firstRay.origin = intersection + 0.0001f * firstRay.direction;
				currDepth++;
				
				finalColor *= glm::clamp(surfColor + phong, 0.0f, 1.0f);
			}
			else{
				finalColor *= glm::clamp(diffuse + phong, 0.0f, 1.0f);
				break;
			}

		}
	
		//output final color
		colors[index] += finalColor;
	}

}


//TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms, bool& clear){
  
  int traceDepth = 10; //determines how many bounces the raytracer traces

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);			//each block has 8 * 8 threads
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  //send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  std::vector<int> lightVec;

  //package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
  
	//store which objects are lights
	if(materials[geoms[i].materialid].emittance > 0)
		lightVec.push_back(i);
  }
  
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);

  //copy materials to memory
  material* cudaMaterials = NULL;
  hipMalloc((void**)&cudaMaterials, numberOfMaterials*sizeof(material));
  hipMemcpy( cudaMaterials, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

  //copy light ID to memeory
  int numLights = lightVec.size();
  int* lightID = new int[numLights];
  for(int i = 0; i <numLights; ++i)
	  lightID[i] = lightVec[i];
  
  int* cudaLights = NULL;
  hipMalloc((void**)&cudaLights, numLights*sizeof(int));
  hipMemcpy( cudaLights, lightID, numLights*sizeof(int), hipMemcpyHostToDevice);
  
  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;
  cam.focalLength = renderCam->focalLengths[frame];
  cam.aperture = renderCam->apertures[frame];

  //cache the first bounce since they're the same for each iteration
  rayBounce* cudaFirstPass = NULL;
  hipMalloc((void**)&cudaFirstPass, cam.resolution.x*cam.resolution.y*sizeof(rayBounce));

  //clear image if camera has been moved
  if(clear){
	  clearImage<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, cudaimage); 
	  clear = false;
  }

  else{
	  //first pass, get rays for first bounce
	  if(iterations == 1) {
		  createRay<<<fullBlocksPerGrid, threadsPerBlock>>>(cam, cudageoms, numberOfGeoms, cudaMaterials, numLights, cudaLights, cudaFirstPass); 
		  
	  }
	  else {
		  raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, cudaimage, cudageoms, numberOfGeoms, 
															cudaMaterials, numLights, cudaLights, cudaFirstPass);
	  }
 
  }

  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage, (float)iterations);

  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);


  //free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  hipFree( cudaMaterials);
  hipFree( cudaLights);
  hipFree( cudaFirstPass);
  delete geomList;
  delete lightID;

  // make certain the kernel has completed
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}

