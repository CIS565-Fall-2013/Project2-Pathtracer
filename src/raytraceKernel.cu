#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>

using namespace glm;

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov){
	vec3 jitter = 2.0f*generateRandomNumberFromThread(resolution, time, x, y);
	float NDCx = ((float)x +jitter.x)/resolution.x;
	float NDCy = ((float)y +jitter.y )/resolution.y;
	
	//float NDCx = ((float)x )/resolution.x;
	//float NDCy = ((float)y )/resolution.y;

	vec3 A = cross(view, up);
	vec3 B = cross(A, view);

	vec3 M = eye+view;
	vec3 V = B * (1.0f/length(B)) * length(view)*tan(radians(fov.y));
	vec3 H = A * (1.0f/length(A)) * length(view)*tan(radians(fov.x));

	vec3 point = M + (2*NDCx -1)*H + (1-2*NDCy)*V;

	ray r;
	r.origin = eye;
	r.direction = normalize(point-eye);
	return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image, float iterations){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;
      color.x = image[index].x/iterations*255.0;
      color.y = image[index].y/iterations*255.0;
      color.z = image[index].z/iterations*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors,
                            staticGeom* geoms, int numberOfGeoms, 
							int numberOfCubes, int numberOfSpheres, material* cudamaterials, int numberOfMaterials, int* cudalights, int numberOfLights){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  if((x<=resolution.x && y<=resolution.y)){

		ray rayFromCamera = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov);

		//find aim point
		vec3 aimPoint = rayFromCamera.origin + cam.focalLength*rayFromCamera.direction;

		//jittered ray (DOF)
		float degOfJitter = 1;
		vec3 jitter = generateRandomNumberFromThread(resolution, time, x, y);
		ray jitteredRay;
		jitteredRay.origin = vec3(rayFromCamera.origin.x+degOfJitter*jitter.x, rayFromCamera.origin.y+degOfJitter*jitter.y, rayFromCamera.origin.z);	
		jitteredRay.direction = normalize(aimPoint-jitteredRay.origin);

		ray currentRay = rayFromCamera; //jitteredRay;
		
		int rayCount = 0;
		vec3 realColor = vec3(0,0,0);
		vec3 accumulReflectiveSurfaceColor = vec3(1,1,1);
		vec3 accumulColor = vec3(0,0,0);

		while(rayCount <= rayDepth){
			float tempLength, closest = 1e26, indexOfRefraction = 0;
			int closestObjectid = -1;
			vec3 tempIntersectionPoint = vec3(0,0,0), tempNormal = vec3(0,0,0), normal = vec3(0,0,0), intersectionPoint = vec3(0,0,0);
			vec3 pixelColor = vec3(0,0,0), objectColor = vec3(0,0,0), specColor = vec3(0,0,0);
			float specExponent = 0, 
			bool isReflective = 0, isRefractive = 0;
			bool inside = false, tempInside = false;

			//input text file must load cubes first before loading spheres

			for (int i = 0; i < numberOfCubes; i++){
				if(geoms[i].type == CUBE){
					tempLength = boxIntersectionTest( geoms[i], currentRay, tempIntersectionPoint, tempNormal, tempInside);
				}

				if (tempLength < closest && tempLength >= 0){
					closest = tempLength;
					normal = tempNormal;
					intersectionPoint = tempIntersectionPoint;
					closestObjectid = i;
					inside = tempInside;
				}
			}

			for(int i = numberOfCubes; i < numberOfGeoms; i++){
				if(geoms[i].type == SPHERE){
					tempLength = sphereIntersectionTest( geoms[i], currentRay, tempIntersectionPoint, tempNormal, tempInside);
				}

				if (tempLength < closest && tempLength >= 0){
					closest = tempLength;
					normal = tempNormal;
					intersectionPoint = tempIntersectionPoint;
					closestObjectid = i;
					inside = tempInside;
				}
			}

			pixelColor = vec3(0,0,0);

			if (closest < 1e26 && closest >= 0){

				objectColor = cudamaterials[geoms[closestObjectid].materialid].color;
				specExponent = cudamaterials[geoms[closestObjectid].materialid].specularExponent;
				specColor = cudamaterials[geoms[closestObjectid].materialid].specularColor;
				isReflective = cudamaterials[geoms[closestObjectid].materialid].hasReflective;
				isRefractive = cudamaterials[geoms[closestObjectid].materialid].hasRefractive;
				indexOfRefraction = cudamaterials[geoms[closestObjectid].materialid].indexOfRefraction;

				vec3 accumulDiffuse = vec3(0,0,0);
				vec3 accumulSpec = vec3(0,0,0);
				vec3 ambient = objectColor;
				vec3 reflectedDir = vec3(0,0,0);
				vec3 refractedDir = vec3(0,0,0);
			
				for (int j = 0; j < numberOfLights; j++){
					if (closestObjectid == cudalights[j]){
						pixelColor = cudamaterials[geoms[closestObjectid].materialid].color;
						colors[index] += pixelColor;
						return;
					}

					vec3 randomPointOnLight;
					if (geoms[cudalights[j]].type == CUBE)
						randomPointOnLight = getRandomPointOnCube(geoms[cudalights[j]],time);
					else if (geoms[cudalights[j]].type == SPHERE)
						randomPointOnLight = getRandomPointOnSphere(geoms[cudalights[j]],time);


					vec3 lightDir = normalize(randomPointOnLight - intersectionPoint);

					//ambient

			
					//diffuse
					vec3 diffuse = dot(normal, lightDir) * cudamaterials[geoms[cudalights[j]].materialid].color * (objectColor);
					//diffuse = vec3(clamp(diffuse.x, 0.0, 1.0), clamp(diffuse.y, 0.0, 1.0), clamp(diffuse.z, 0.0, 1.0));


					vec3 specular = vec3(0,0,0);
					reflectedDir = currentRay.direction - vec3(2*vec4(normal*(dot(currentRay.direction,normal)),0));
					reflectedDir = normalize(reflectedDir);
					//specular phong lighting
					if (specExponent > 0){
						//vec3 reflectedDir = lightDir -  vec3(2*vec4(normal*(dot(lightDir,normal)),0));
						float D = dot(reflectedDir, lightDir);
						if (D < 0) D = 0;
						specular = specColor*pow(D, specExponent);
					}

					//shadows, see if there is an object between light and pixel.
					ray pointToLight; 
					pointToLight.origin = intersectionPoint;
					pointToLight.direction = lightDir;
					float lengthFromPointToLight;
					if (geoms[cudalights[j]].type == CUBE)
						lengthFromPointToLight = boxIntersectionTest( geoms[cudalights[j]], pointToLight, tempIntersectionPoint, tempNormal, tempInside);
					else if (geoms[cudalights[j]].type == SPHERE)
						lengthFromPointToLight = sphereIntersectionTest( geoms[cudalights[j]], pointToLight, tempIntersectionPoint, tempNormal, tempInside);
					tempLength = 1e26;
					int occluded = -1;
					for (int i = 0; i < numberOfGeoms; i++){
						if (i != closestObjectid){
							if(geoms[i].type == CUBE){
								tempLength = boxIntersectionTest( geoms[i], pointToLight, tempIntersectionPoint, tempNormal, tempInside);
							}else{
								tempLength = sphereIntersectionTest(geoms[i], pointToLight, tempIntersectionPoint, tempNormal, tempInside);
							}

							if (tempLength < lengthFromPointToLight && tempLength != -1){
								occluded = i;
								i = numberOfGeoms; 
							}
						}
					}

					//apply shadow, make darker
					bool hitLight = false;
					for (int x = 0; x < numberOfLights; x++){
						if (occluded == cudalights[x]){
							hitLight = true;
							break;
						}
					}
					if (occluded != -1 && !hitLight){
						//diffuse *= .2f;
						//specular *= .2f;
						diffuse = vec3(0,0,0);
						specular = vec3(0,0,0);
					}

					accumulDiffuse += diffuse;
					accumulDiffuse = clamp(accumulDiffuse, vec3(0,0,0), objectColor);
					accumulSpec += specular;
					accumulSpec = clamp(accumulSpec, vec3(0,0,0), vec3(1,1,1));

				}//for loop

				if (specExponent > 0){
					accumulColor += .5f*accumulDiffuse + .4f*accumulSpec + .1f*ambient;
				}else{
					accumulColor += accumulDiffuse;
				}
				accumulColor = clamp(accumulColor, vec3(0,0,0), vec3(1,1,1));

				float n1 = 0, n2 = 0;
				float costheta_i = 0; float costheta_t = 0;
				float sin2theta_t = 0;
				float R = 0;
				bool TIR = false;
				float schlicksR = 0;
				float random = 0;

				if (isRefractive){

					//graphics.stanford.edu/courses/cs148-10-summer/docs/2006--degreve--reflection_refraction.pdf

					if (inside){
						n1 = indexOfRefraction;
						n2 = 1.0f;
						normal = -normal;
					}else{
						n1 = 1.0f;
						n2 = indexOfRefraction;
					}

					costheta_i = glm::dot(-1.0f*currentRay.direction, normal);
					sin2theta_t = pow(n1/n2,2)*(1-pow(costheta_i,2));
					R = pow((n1-n2)/(n1+n2),2);
					if (sin2theta_t > 1){
						TIR = true;
					}else{
						costheta_t = sqrt(1-sin2theta_t);
						refractedDir = (n1/n2)*currentRay.direction + ((n1/n2)*costheta_i - sqrt(1-sin2theta_t))*normal;
					}

					if (n1 <= n2){
						schlicksR = R + (1-R)*pow(1-costheta_i,5);
					}else if (n1 > n2 && !TIR){
						schlicksR = R + (1-R)*pow(1-costheta_t,5);
					}else{
						schlicksR = 1;
					}
  
					thrust::default_random_engine rng(hash((x + (y * resolution.x))*time));
					thrust::uniform_real_distribution<float> u01(0,1);

					random = (float) u01(rng);
					
					currentRay.origin = intersectionPoint+0.01f*refractedDir;
					currentRay.direction = refractedDir;
					
					if (random <= schlicksR){
						currentRay.origin = intersectionPoint+0.0001f*reflectedDir;
						currentRay.direction = reflectedDir;
					}

					accumulReflectiveSurfaceColor *= objectColor; //accumulColor;

				}else if (isReflective){
					currentRay.origin = intersectionPoint+0.0001f*reflectedDir;
					currentRay.direction = reflectedDir;
					accumulReflectiveSurfaceColor *= objectColor; //accumulColor;
				}else{
					rayCount = rayDepth;
				}

			}//if intersects with anything

			rayCount++;

		}//while loop

		realColor = accumulReflectiveSurfaceColor*accumulColor;
		//realColor = accumulColor;
		colors[index] += realColor;
   }
}

//TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms, int numberOfCubes, int numberOfSpheres, bool cameraMoved){
  
  int traceDepth = 5; //determines how many bounces the raytracer traces
  std::vector<int> lightsid;

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));
  
  //send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  //package geometry and materials and sent to GPU
  staticGeom* geomList = new staticGeom[numberOfGeoms];
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
	if (materials[newStaticGeom.materialid].emittance > 0)
		lightsid.push_back(i);
  }

  int* lightsList = new int[lightsid.size()];
  for (int i = 0; i < lightsid.size(); i++){
	  lightsList[i] = lightsid[i];
  }

     
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);

  material* cudamaterials = NULL;
  hipMalloc((void**)&cudamaterials, numberOfMaterials*sizeof(material));
  hipMemcpy( cudamaterials, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

  int* cudalights = NULL;
  hipMalloc((void**)&cudalights, lightsid.size()*sizeof(int));
  hipMemcpy( cudalights, lightsList, lightsid.size()*sizeof(int), hipMemcpyHostToDevice);
    
  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;
  cam.focalLength = renderCam->focalLengths[frame];

  //clear image
  if (cameraMoved)
	clearImage<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution,cudaimage);

  if (numberOfGeoms != numberOfCubes+numberOfSpheres){
	  std::cout<<"ERROR numberOfGeoms != numberOfCubes+numberOfSpheres"<<std::endl;
	  std::cout<<numberOfGeoms<<", "<<numberOfCubes<<", "<<numberOfSpheres<<std::endl;
  }

  //kernel launches
  raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth, 
													cudaimage, cudageoms, numberOfGeoms, numberOfCubes, numberOfSpheres, cudamaterials, numberOfMaterials, cudalights, lightsid.size());

  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage, (float)iterations);

  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

  //free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
  delete geomList;

  // make certain the kernel has completed
  hipDeviceSynchronize();
  checkCUDAError("Kernel failed!");
}
