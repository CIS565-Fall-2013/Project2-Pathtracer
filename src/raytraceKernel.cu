#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>
#include <set>

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

#define MAXDEPTH 10 // max raytrace depth

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//---------------------------------------------
//--------------Helper functions---------------
//---------------------------------------------

// Returns true if every component of a is greater than the corresponding component of b
__host__ __device__ bool componentCompare(glm::vec3 a, glm::vec3 b) {
	return (a[0] > b[0] && a[1] > b[1] && a[2] > b[2]);
}

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y,glm::vec3 eye,
																								glm::vec3 view, glm::vec3 up, glm::vec2 fov, float focal, float aperture){
  ray r;
  r.origin = eye;

	// values for computing ray direction
  float phi = glm::radians(fov.y);
	float theta = glm::radians(fov.x);
	glm::vec3 A = glm::normalize(glm::cross(view, up));
	glm::vec3 B = glm::normalize(glm::cross(A, view));
	glm::vec3 M = eye + view;
	glm::vec3 V = B * glm::length(view) * tan(phi);
	glm::vec3 H = A * glm::length(view) * tan(theta);

	// super sampling for anti-aliasing
	thrust::default_random_engine rng(hash(time));
	thrust::uniform_real_distribution<float> u01(0, 1);
	float fx = x + (float)u01(rng);
	float fy = y + (float)u01(rng);

	glm::vec3 P = M + (2*fx/(resolution.x-1)-1) * H + (2*(1-fy/(resolution.y-1))-1) * V;
	r.direction = glm::normalize(P-eye);

	if (abs(focal+1) > THRESHOLD) {
		// for depth of field
		// get the intersection with the focal plane
		glm::vec3 pointOnFocalPlane = eye + r.direction * focal;

		// jitter sample position
		thrust::uniform_real_distribution<float> u02(-aperture/2, aperture/2);
		r.origin += A * u02(rng) + B * u02(rng);
		r.direction = glm::normalize(pointOnFocalPlane - r.origin);
	}

  return r;
}

// Get the reflected ray direction from ray direction and normal
__host__ __device__ glm::vec3 getReflectedRay(glm::vec3 d, glm::vec3 n) {
	glm::vec3 VR; // reflected ray direction
	if (glm::length(-d - n) < THRESHOLD) {
		VR = n;
	}
	else if (abs(glm::dot(-d, n)) < THRESHOLD) {
		VR = d;
	}
	else {
		VR = glm::normalize(d - 2.0f * glm::dot(d, n) * n);
	}
	return VR;
}

// Get the refracted ray direction from ray direction, normal and index of refraction (IOR)
__host__ __device__ glm::vec3 getRefractedRay(glm::vec3 d, glm::vec3 n, float IOR) {
	glm::vec3 VT; // refracted ray direction
	float t = 1 / IOR;
	float base = 1 - t * t * (1 - pow(glm::dot(n, d), 2));
	if (base < 0) {
		 VT = glm::vec3(0, 0, 0);
	}
	else {
		VT = (-t * glm::dot(n, d) - sqrt(base)) * n + t * d; // refracted ray
		VT = glm::normalize(VT);
	}
	return VT;
}

// Determine if the randomly generated ray is a refracted ray or a reflected ray
__host__ __device__  bool isRefractedRay(float randomSeed, float IOR, glm::vec3 d, glm::vec3 n, glm::vec3 t) {
	float rpar = (IOR * glm::dot(n, d) - glm::dot(n, t)) / (IOR * glm::dot(n, d) + glm::dot(n, t));
	float rperp = (glm::dot(n, d) - IOR * glm::dot(n, t)) / (glm::dot(n, d) + IOR * glm::dot(n, t));

	// compute proportion of the light reflected
	float fr = 0.5 * (rpar * rpar + rperp * rperp);

	// determine if ray is reflected according to the proportion
	thrust::default_random_engine rng(hash(randomSeed));
	thrust::uniform_real_distribution<float> u01(0,1);
	if (u01(rng) <= fr) {
		return false;
	}
	else {
		return true;
	}
}

// Decide if the intersection point is in the shadow of a light
__host__ __device__ bool isInShadow(staticGeom* geoms, int numberOfGeoms, int lightIdx, glm::vec3 intersection,
																		int pixelIdx, float time, glm::vec3& L /*out*/) {
	// generate shadow feeler
	glm::vec3 pointOnLight = getRandomPointOnGeom(geoms[lightIdx], pixelIdx * time); // area light
	float distToLight = glm::distance(intersection, pointOnLight);
	L = glm::normalize(intersection - pointOnLight); // direction from light to point
	ray shadowFeeler;
	shadowFeeler.origin = intersection + (-L) * (float)THRESHOLD;
	shadowFeeler.direction = -L;
	
	// find out if the shadow feeler intersects other objects
	bool shadow = false;
	for (int j=0; j<numberOfGeoms; ++j) {
		if (j != lightIdx) {
			glm::vec3 intersection, normal;
			float dist = geomIntersectionTest(geoms[j], shadowFeeler, intersection, normal);
			if (abs(dist+1) > THRESHOLD && dist < distToLight) {
				shadow = true;
				break;
			}
		}
	}
	return shadow;
}

//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, globalAttributes globalAttr, int rayDepth, glm::vec3* colors,
														staticGeom* geoms, int numberOfGeoms, material* materials, int* lightIds, int numberOfLights){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

	glm::vec3 color;

  if((x<=resolution.x && y<=resolution.y)){
		ray r = raycastFromCameraKernel(resolution, time, x, y, cam.position, cam.view, cam.up, cam.fov, cam.focal, cam.aperture);
		glm::vec3 baseColor(1, 1, 1);

		for (int iteration=0; iteration<MAXDEPTH; ++iteration) {
			glm::vec3 minIntersection, minNormal; // closest intersection point and the normal at that point
			int minIdx = getClosestIntersection(geoms, numberOfGeoms, r, minIntersection, minNormal);

			if (minIdx != -1) {
				material mtl = materials[geoms[minIdx].materialid]; // does caching make it faster?
				//TODO: MAKE THIS BRANCH MORE EFFICIENT
				if (mtl.emittance > THRESHOLD) { // light
					color = glm::clamp(mtl.color * mtl.emittance, glm::vec3(0, 0, 0), glm::vec3(1, 1, 1));
				}
				else {
					if (mtl.hasReflective < THRESHOLD && mtl.hasRefractive < THRESHOLD) { // use phong shading model
						glm::vec3 ambient = globalAttr.ambient * mtl.color;
						ambient = glm::clamp(ambient, glm::vec3(0, 0, 0), glm::vec3(1, 1, 1));
						glm::vec3 diffuse(0, 0, 0);
						glm::vec3 specular(0, 0, 0);

						if (glm::dot(r.direction, minNormal) > 0) { // reverse normal if we are inside the object
							minNormal *= -1;
						}
						
						for (int i=0; i<numberOfLights; ++i) {
							glm::vec3 L;
							if (!isInShadow(geoms, numberOfGeoms, lightIds[i], minIntersection, index, time, L)) {
								material lightMtl = materials[geoms[lightIds[i]].materialid];
								glm::vec3 lightColor = glm::clamp(lightMtl.color * lightMtl.emittance, glm::vec3(0, 0, 0), glm::vec3(1, 1, 1));
								
								// compute diffuse color
								diffuse += lightColor * mtl.color * glm::clamp(glm::dot(-L, minNormal), 0.0f, 1.0f);
								if (componentCompare(diffuse, mtl.color)) {
									break;
								}

								if (mtl.specularExponent > THRESHOLD) {
									// compute specular color
									glm::vec3 LR = getReflectedRay(L, minNormal); 
									specular += lightColor * mtl.specularColor * pow(glm::clamp(glm::dot(LR, -r.direction), 0.0f, 1.0f), mtl.specularExponent);
									if (componentCompare(specular, mtl.specularColor)) {
										break;
									}
								}
							}
						}
						diffuse = glm::clamp(diffuse, glm::vec3(0, 0, 0), mtl.color);
						specular = glm::clamp(specular, glm::vec3(0, 0, 0), mtl.specularColor);
						color = glm::clamp(globalAttr.Ka * ambient + globalAttr.Kd * diffuse + globalAttr.Ks * specular, glm::vec3(0, 0, 0), glm::vec3(1, 1, 1));
						color = color * baseColor;
						break;
					}
					else {
						float IOR = mtl.indexOfRefraction;
						if (glm::dot(r.direction, minNormal) > 0) { // reverse normal and index of refraction if we are inside the object
							minNormal *= -1;
							IOR = 1/(IOR + THRESHOLD);
						}
						if (mtl.hasRefractive > THRESHOLD) { // if the surface has refraction
							glm::vec3 VT = getRefractedRay(r.direction, minNormal, IOR);
							if (glm::length(VT) > THRESHOLD && (mtl.hasReflective < THRESHOLD || isRefractedRay(index*time, IOR, r.direction, minNormal, VT))) {
								r.direction = VT;
								r.origin = minIntersection + VT * (float)THRESHOLD;
								//baseColor *= mtl.color;
								continue;
							}
						}
						// if the surface only has reflection
						glm::vec3 VR = getReflectedRay(r.direction, minNormal);
						r.origin = minIntersection + VR * (float)THRESHOLD;
						r.direction = VR;
						baseColor *= mtl.color;
					}
				}
			}
			else {
				color = glm::vec3(0, 0, 0);
			}
		}
  }
	colors[index] = (colors[index] * (time-1) + color)/time;
}

//TODO: FINISH THIS FUNCTION
// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, globalAttributes globalAttr, int frame, int iterations,
											material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms){
  
  int traceDepth = 1; //determines how many bounces the raytracer traces

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));  //send image to GPU
  glm::vec3* cudaimage = NULL;
  hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
  hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
  
  // keep track of the IDs of light materials
	std::set<int> lightMtlIds;
	for (int i=0; i<numberOfMaterials; ++i) {
		if (materials[i].emittance > 0) {
			lightMtlIds.insert(i);
		}
	}
	
	//package geometry and materials and sent to GPU
	// keep track of the IDs of light geometries
  staticGeom* geomList = new staticGeom[numberOfGeoms];
	std::vector<int> lightIds;
  for(int i=0; i<numberOfGeoms; i++){
    staticGeom newStaticGeom;
    newStaticGeom.type = geoms[i].type;
    newStaticGeom.materialid = geoms[i].materialid;
		if (lightMtlIds.find(newStaticGeom.materialid) != lightMtlIds.end()) {
			lightIds.push_back(i);
		}
    newStaticGeom.translation = geoms[i].translations[frame];
    newStaticGeom.rotation = geoms[i].rotations[frame];
    newStaticGeom.scale = geoms[i].scales[frame];
    newStaticGeom.transform = geoms[i].transforms[frame];
    newStaticGeom.inverseTransform = geoms[i].inverseTransforms[frame];
    geomList[i] = newStaticGeom;
  }
	int* lightIdList = new int[lightIds.size()];
	for (int i=0; i<lightIds.size(); i++) {
		lightIdList[i] = lightIds[i];
	}
  
  staticGeom* cudageoms = NULL;
  hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
  hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);

	// copy materials to CUDA
	material* cudamtls = NULL;
	hipMalloc((void**)&cudamtls, numberOfMaterials*sizeof(material));
  hipMemcpy( cudamtls, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

	// copy lights to CUDA
	int* cudalights = NULL;
	hipMalloc((void**)&cudalights, lightIds.size()*sizeof(int));
	hipMemcpy( cudalights, lightIdList, lightIds.size()*sizeof(int), hipMemcpyHostToDevice);
  
  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.fov = renderCam->fov;
	cam.focal = renderCam->focal;
	cam.aperture = renderCam->aperture;

  //kernel launches
	raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, globalAttr, traceDepth, cudaimage,
		cudageoms, numberOfGeoms, cudamtls, cudalights, lightIds.size());

  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage);

  //retrieve image from GPU
  hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);

  //free up stuff, or else we'll leak memory like a madman
  hipFree( cudaimage );
  hipFree( cudageoms );
	hipFree( cudamtls );
	hipFree( cudalights );
  delete geomList;
	delete lightIdList;

  // make certain the kernel has completed
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}
